#include "hip/hip_runtime.h"
//
// Author: Bryan Riel
// Copyright 2018
//

#include "gpuGeometry.h"
#include "../hip/hip_runtime_api.h"
#include "isce/cuda/core/gpuPixel.h"

using isce::cuda::core::gpuLinAlg;

/** @param[in] pixel gpuPixel object
 * @param[in] TCNbasis Geocentric TCN basis corresponding to pixel
 * @param[in] state gpuStateVector object
 * @param[in] ellipsoid gpuEllipsoid object
 * @param[in] demInterp gpuDEMInterpolator object
 * @param[out] targetLLH output Lon/Lat/Hae corresponding to pixel
 * @param[in] side +1 for left and -1 for right
 * @param[in] threshold Distance threshold for convergence
 * @param[in] maxIter Number of primary iterations
 * @param[in] extraIter Number of secondary iterations
 *
 * This is the elementary device-side transformation from radar geometry to map geometry. The transformation is applicable for a single slant range and azimuth time. The slant range and Doppler information are encapsulated in the Pixel object, so this function can work for both zero and native Doppler geometries. The azimuth time information is encapsulated in the TCNbasis and StateVector of the platform. For algorithmic details, see \ref overview_geometry "geometry overview".*/
CUDA_DEV
int isce::cuda::geometry::
rdr2geo(const isce::cuda::core::gpuPixel & pixel,
        const isce::cuda::core::gpuBasis & TCNbasis,
        const isce::cuda::core::gpuStateVector & state,
        const isce::cuda::core::gpuEllipsoid & ellipsoid,
        const gpuDEMInterpolator & demInterp,
        double * targetLLH,
        int side, double threshold, int maxIter, int extraIter) {

    // Initialization
    double targetVec[3], targetLLH_old[3], targetVec_old[3],
           lookVec[3], delta[3], delta_temp[3], vhat[3];

    // Compute normalized velocity
    gpuLinAlg::unitVec(state.velocity, vhat);

    // Unpack TCN basis vectors to pointers
    const double * that = TCNbasis.x0;
    const double * chat = TCNbasis.x1;
    const double * nhat = TCNbasis.x2;

    // Pre-compute TCN vector products
    const double ndotv = nhat[0]*vhat[0] + nhat[1]*vhat[1] + nhat[2]*vhat[2];
    const double vdott = vhat[0]*that[0] + vhat[1]*that[1] + vhat[2]*that[2];

    // Compute major and minor axes of ellipsoid
    const double major = ellipsoid.a;
    const double minor = major * std::sqrt(1.0 - ellipsoid.e2);

    // Set up orthonormal system right below satellite
    const double satDist = gpuLinAlg::norm(state.position);
    const double eta = 1.0 / std::sqrt(
        std::pow(state.position[0] / major, 2) +
        std::pow(state.position[1] / major, 2) +
        std::pow(state.position[2] / minor, 2)
    );
    const double radius = eta * satDist;
    const double hgt = (1.0 - eta) * satDist;

    // Iterate
    int converged = 0;
    double zrdr = targetLLH[2];
    for (int i = 0; i < (maxIter + extraIter); ++i) {

        // Near nadir test
        if ((hgt - zrdr) >= pixel.range())
            break;

        // Cache the previous solution
        for (int k = 0; k < 3; ++k) {
            targetLLH_old[k] = targetLLH[k];
        }

        // Compute angles
        const double a = satDist;
        const double b = radius + zrdr;
        const double costheta = 0.5 * (a / pixel.range() + pixel.range() / a
                              - (b/a) * (b/pixel.range()));
        const double sintheta = std::sqrt(1.0 - costheta*costheta);

        // Compute TCN scale factors
        const double gamma = pixel.range() * costheta;
        const double alpha = (pixel.dopfact() - gamma * ndotv) / vdott;
        const double beta = -side * std::sqrt(std::pow(pixel.range(), 2)
                                            * std::pow(sintheta, 2)
                                            - std::pow(alpha, 2));
    
        // Compute vector from satellite to ground
        gpuLinAlg::linComb(alpha, that, beta, chat, delta_temp);
        gpuLinAlg::linComb(1.0, delta_temp, gamma, nhat, delta);
        gpuLinAlg::linComb(1.0, state.position, 1.0, delta, targetVec);

        // Compute LLH of ground point
        ellipsoid.xyzToLonLat(targetVec, targetLLH);

        // Interpolate DEM at current lat/lon point
        targetLLH[2] = demInterp.interpolateLonLat(targetLLH[0], targetLLH[1]);

        // Convert back to XYZ with interpolated height
        ellipsoid.lonLatToXyz(targetLLH, targetVec);
        // Compute updated target height
        zrdr = gpuLinAlg::norm(targetVec) - radius;

        // Check convergence
        gpuLinAlg::linComb(1.0, state.position, -1.0, targetVec, lookVec);
        const double rdiff = pixel.range() - gpuLinAlg::norm(lookVec);
        if (std::abs(rdiff) < threshold) {
            converged = 1;
            break;
        // May need to perform extra iterations
        } else if (i > maxIter) {
            // XYZ position of old solution
            ellipsoid.lonLatToXyz(targetLLH_old, targetVec_old);
            // XYZ position of updated solution
            for (int idx = 0; idx < 3; ++idx)
                targetVec[idx] = 0.5 * (targetVec_old[idx] + targetVec[idx]);
            // Repopulate lat, lon, z
            ellipsoid.xyzToLonLat(targetVec, targetLLH);
            // Compute updated target height
            zrdr = gpuLinAlg::norm(targetVec) - radius;
        }
    }

    // ----- Final computation: output points exactly at range pixel if converged

    // Compute angles
    const double a = satDist;
    const double b = radius + zrdr;
    const double costheta = 0.5 * (a / pixel.range() + pixel.range() / a
                          - (b/a) * (b/pixel.range()));
    const double sintheta = std::sqrt(1.0 - costheta*costheta);

    // Compute TCN scale factors
    const double gamma = pixel.range() * costheta;
    const double alpha = (pixel.dopfact() - gamma * ndotv) / vdott;
    const double beta = -side * std::sqrt(std::pow(pixel.range(), 2)
                                        * std::pow(sintheta, 2)
                                        - std::pow(alpha, 2));

    // Compute vector from satellite to ground
    gpuLinAlg::linComb(alpha, that, beta, chat, delta_temp);
    gpuLinAlg::linComb(1.0, delta_temp, gamma, nhat, delta);
    gpuLinAlg::linComb(1.0, state.position, 1.0, delta, targetVec);

    // Compute LLH of ground point
    ellipsoid.xyzToLonLat(targetVec, targetLLH);

    // Return convergence flag
    return converged;
}

// Utility function to compute geocentric tcn basis from state vector
__device__ void geocentricTCN(const double* pos, const double* vel,
                              isce::cuda::core::gpuBasis& basis) {
    double t_hat[3], c_hat[3], n_hat[3], temp[3];
    using isce::cuda::core::gpuLinAlg;
    // Compute basis vectors
    gpuLinAlg::unitVec(pos, n_hat);
    gpuLinAlg::scale(n_hat, -1.);
    gpuLinAlg::cross(n_hat, vel, temp);
    gpuLinAlg::unitVec(temp, c_hat);
    gpuLinAlg::cross(c_hat, n_hat, temp);
    gpuLinAlg::unitVec(temp, t_hat);
    // Store in basis object
    for (int i = 0; i < 3; i++) {
        basis.x0[i] = t_hat[i];
        basis.x1[i] = c_hat[i];
        basis.x2[i] = n_hat[i];
    }
}

/*
 * This is a helper function for calling the more primitive version of rdr2geo.
 * For more information, see its CPU equivalent in isce/geometry/geometry.h.
 * This GPU version is simplified since it cannot perform error checking.
 */
__device__ int isce::cuda::geometry::rdr2geo(
        double aztime, double slant_range, double doppler,
        const isce::cuda::core::gpuOrbit& orbit,
        const isce::cuda::core::gpuEllipsoid& ellipsoid,
        const isce::cuda::geometry::gpuDEMInterpolator& dem_interp,
        double* target_llh, double wvl, int side, double threshold,
        int max_iter, int extra_iter) {

    /*
     * Interpolate Orbit to azimuth time, compute TCN basis,
     * and estimate geographic coordinates.
     */

    // Interpolate orbit to get state vector
    double pos[3], vel[3];
    orbit.interpolateWGS84Orbit(aztime, pos, vel);

    // Set up geocentric TCN basis
    isce::cuda::core::gpuBasis tcn_basis;
    geocentricTCN(pos, vel, tcn_basis);

    // Compute satellite velocity magnitude
    const double vmag = isce::cuda::core::gpuLinAlg::norm(vel);

    // Compute Doppler factor
    const double dopfact = 0.5 * wvl * doppler * slant_range / vmag;

    // Wrap range and Doppler factor in a Pixel object
    isce::cuda::core::gpuPixel pixel(slant_range, dopfact, 0);

    isce::cuda::core::gpuStateVector state;
    for (int i = 0; i < 3; i++) {
        state.position[i] = pos[i];
        state.velocity[i] = vel[i];
    }

    // Finally, call rdr2geo
    return rdr2geo(pixel, tcn_basis, state, ellipsoid, dem_interp,
                   target_llh, side, threshold, max_iter, extra_iter);
}

/** @param[in] inputLLH Lon/Lat/Hae of target of interest
 * @param[in] ellipsoid gpuEllipsoid object
 * @param[in] orbit gpuOrbit object
 * @param[in] doppler gpuLUT1d Doppler model
 * @param[in] mode  gpuImageMode object
 * @param[out] aztime azimuth time of inputLLH w.r.t reference epoch of the orbit
 * @param[out] slantRange slant range to inputLLH
 * @param[in] threshold azimuth time convergence threshold in seconds
 * @param[in] maxIter Maximum number of Newton-Raphson iterations
 * @param[in] deltaRange step size used for computing derivative of doppler
 *
 * This is the elementary device-side transformation from map geometry to radar geometry. The transformation is applicable for a single lon/lat/h coordinate (i.e., a single point target). For algorithmic details, see \ref overview_geometry "geometry overview".*/
CUDA_DEV
int isce::cuda::geometry::
geo2rdr(double * inputLLH,
        const isce::cuda::core::gpuEllipsoid & ellipsoid,
        const isce::cuda::core::gpuOrbit & orbit,
        const isce::cuda::core::gpuLUT1d<double> & doppler,
        const isce::cuda::product::gpuImageMode & mode,
        double * aztime_result, double * slantRange_result,
        double threshold, int maxIter, double deltaRange) {

    // Cartesian type local variables
    double inputXYZ[3], satpos[3], satvel[3], dr[3];
    // Temp local variables for results
    double aztime, slantRange;

    // Convert LLH to XYZ
    ellipsoid.lonLatToXyz(inputLLH, inputXYZ);

    // Pre-compute scale factor for doppler
    const double dopscale = 0.5 * mode.wavelength();

    // Use mid-orbit epoch as initial guess
    aztime = orbit.UTCtime[orbit.nVectors / 2];
    
    // Begin iterations
    int converged = 0;
    double slantRange_old = 0.0;
    for (int i = 0; i < maxIter; ++i) {

        // Interpolate the orbit to current estimate of azimuth time
        orbit.interpolateWGS84Orbit(aztime, satpos, satvel);

        // Compute slant range from satellite to ground point
        gpuLinAlg::linComb(1.0, inputXYZ, -1.0, satpos, dr);
        slantRange = gpuLinAlg::norm(dr);
        // Check convergence
        if (std::abs(slantRange - slantRange_old) < threshold) {
            converged = 1;
            *slantRange_result = slantRange;
            *aztime_result = aztime;
            return converged;
        } else {
            slantRange_old = slantRange;
        }

        // Compute doppler
        const double dopfact = gpuLinAlg::dot(dr, satvel);
        const double fdop = doppler.eval(slantRange) * dopscale;
        // Use forward difference to compute doppler derivative
        const double fdopder = (doppler.eval(slantRange + deltaRange) * dopscale - fdop)
                             / deltaRange;

        // Evaluate cost function and its derivative
        const double fn = dopfact - fdop * slantRange;
        const double c1 = -1.0 * gpuLinAlg::dot(satvel, satvel);
        const double c2 = (fdop / slantRange) + fdopder;
        const double fnprime = c1 + c2 * dopfact;

        // Update guess for azimuth time
        aztime -= fn / fnprime;
    }

    // If we reach this point, no convergence for specified threshold
    *slantRange_result = slantRange;
    *aztime_result = aztime;
    return converged;

}

// Create ProjectionBase pointer on the device (meant to be run by a single thread)
__global__
void
createProjection(isce::cuda::core::ProjectionBase ** proj, int epsgCode) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        (*proj) = isce::cuda::core::createProj(epsgCode);
    }
}

// Delete ProjectionBase pointer on the device (meant to be run by a single thread)
__global__
void
deleteProjection(isce::cuda::core::ProjectionBase ** proj) {
    delete *proj;
}

// Helper kernel to call device-side rdr2geo
__global__
void rdr2geo_d(const isce::cuda::core::gpuPixel pixel,
               const isce::cuda::core::gpuBasis TCNbasis,
               const isce::cuda::core::gpuStateVector state,
               const isce::cuda::core::gpuEllipsoid ellipsoid,
               isce::cuda::geometry::gpuDEMInterpolator demInterp,
               double * targetLLH,
               int side, double threshold, int maxIter, int extraIter,
               int *resultcode) {

    // Call device function
    *resultcode = isce::cuda::geometry::rdr2geo(
        pixel, TCNbasis, state, ellipsoid, demInterp, targetLLH, side,
        threshold, maxIter, extraIter
    );

}

// Host radar->geo to test underlying functions in a single-threaded context
CUDA_HOST
int isce::cuda::geometry::
rdr2geo_h(const isce::core::Pixel & pixel,
          const isce::core::Basis & basis,
          const isce::core::StateVector & state,
          const isce::core::Ellipsoid & ellipsoid,
          isce::geometry::DEMInterpolator & demInterp,
          cartesian_t & llh,
          int side, double threshold, int maxIter, int extraIter) {

    // Make GPU objects
    isce::cuda::core::gpuPixel gpu_pixel(pixel);
    isce::cuda::core::gpuBasis gpu_basis(basis);
    isce::cuda::core::gpuStateVector gpu_state(state);
    isce::cuda::core::gpuEllipsoid gpu_ellps(ellipsoid);
    isce::cuda::geometry::gpuDEMInterpolator gpu_demInterp(demInterp);
        
    // Allocate device memory
    double * llh_d;
    int * resultcode_d;
    hipMalloc((double **) &llh_d, 3*sizeof(double));
    hipMalloc((int **) &resultcode_d, sizeof(int));

    // Copy initial values
    hipMemcpy(llh_d, llh.data(), 3*sizeof(double), hipMemcpyHostToDevice);

    // DEM interpolator initializes its projection and interpolator
    gpu_demInterp.initProjInterp();
    
    // Run the rdr2geo on the GPU
    dim3 grid(1), block(1);
    rdr2geo_d<<<grid, block>>>(gpu_pixel, gpu_basis, gpu_state, gpu_ellps,
                               gpu_demInterp, llh_d, side, threshold, maxIter,
                               extraIter, resultcode_d);

    // Check for any kernel errors
    checkCudaErrors(hipPeekAtLastError());

    // Delete projection pointer on device
    gpu_demInterp.finalizeProjInterp();

    // Copy the resulting llh back to the CPU
    int resultcode;
    checkCudaErrors(hipMemcpy(llh.data(), llh_d, 3*sizeof(double), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(&resultcode, resultcode_d, sizeof(int), hipMemcpyDeviceToHost));

    // Free memory
    checkCudaErrors(hipFree(llh_d));
    checkCudaErrors(hipFree(resultcode_d));

    // Return result code
    return resultcode;
}

// Helper kernel to call device-side geo2rdr
__global__
void geo2rdr_d(double * llh,
               isce::cuda::core::gpuEllipsoid ellps,
               isce::cuda::core::gpuOrbit orbit,
               isce::cuda::core::gpuLUT1d<double> doppler,
               isce::cuda::product::gpuImageMode mode,
               double * aztime, double * slantRange,
               double threshold, int maxIter, double deltaRange,
               int *resultcode) {

    // Call device function
    *resultcode = isce::cuda::geometry::geo2rdr(
        llh, ellps, orbit, doppler, mode, aztime, slantRange, threshold,
        maxIter, deltaRange
    );
                          
}

// Host geo->radar to test underlying functions in a single-threaded context
CUDA_HOST
int isce::cuda::geometry::
geo2rdr_h(const cartesian_t & llh,
          const isce::core::Ellipsoid & ellps,
          const isce::core::Orbit & orbit,
          const isce::core::LUT1d<double> & doppler,
          const isce::product::ImageMode & mode,
          double & aztime, double & slantRange,
          double threshold, int maxIter, double deltaRange) {

    // Make GPU objects
    isce::cuda::core::gpuEllipsoid gpu_ellps(ellps);
    isce::cuda::core::gpuOrbit gpu_orbit(orbit);
    isce::cuda::core::gpuLUT1d<double> gpu_doppler(doppler);
    isce::cuda::product::gpuImageMode gpu_mode(mode);

    // Allocate necessary device memory
    double *llh_d, *aztime_d, *slantRange_d;
    int *resultcode_d;
    hipMalloc((double **) &llh_d, 3*sizeof(double));
    hipMalloc((double **) &aztime_d, sizeof(double));
    hipMalloc((double **) &slantRange_d, sizeof(double));
    hipMalloc((int **) &resultcode_d, sizeof(int));

    // Copy input values
    hipMemcpy(llh_d, llh.data(), 3*sizeof(double), hipMemcpyHostToDevice);

    // Run geo2rdr on the GPU
    dim3 grid(1), block(1);
    geo2rdr_d<<<grid, block>>>(llh_d, gpu_ellps, gpu_orbit, gpu_doppler, gpu_mode,
                               aztime_d, slantRange_d, threshold, maxIter, deltaRange,
                               resultcode_d);

    // Copy results to CPU and return any error code
    int resultcode;
    hipMemcpy(&aztime, aztime_d, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&slantRange, slantRange_d, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&resultcode, resultcode_d, sizeof(int), hipMemcpyDeviceToHost);

    // Free memory
    hipFree(llh_d);
    hipFree(aztime_d);
    hipFree(slantRange_d);
    hipFree(resultcode_d);

    // Return error code
    return resultcode;
}
