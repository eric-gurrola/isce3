#include "hip/hip_runtime.h"
#include <complex>
#include <cmath>
#include <iostream>
#include <iomanip>
#include <string>
#include <cstdio>
#include <fstream>
#include <complex>
#include <ctime>
#include <cstring>

#include "isce/core/Constants.h"
#include "isce/core/DateTime.h"
#include "isce/core/Ellipsoid.h"
#include "isce/core/Peg.h"
#include "isce/core/Pegtrans.h"
#include "isce/geometry/RTC.h"
#include "isce/geometry/Topo.h"

#include "../hip/hip_runtime_api.h"
#include "isce/cuda/core/gpuEllipsoid.h"
#include "isce/cuda/core/gpuLinAlg.h"
#include "isce/cuda/core/gpuLUT1d.h"
#include "isce/cuda/core/gpuOrbit.h"
#include "isce/cuda/geometry/gpuGeometry.h"
#include "isce/cuda/geometry/gpuDEMInterpolator.h"
#include "isce/cuda/product/gpuImageMode.h"

__constant__ double start, r0, pixazm, dr;
__constant__ float xbound, ybound;

// Function to compute normal vector to a plane given three points
__device__
void computePlaneNormal(double* nhat, const double* p1,
                                      const double* p2,
                                      const double* p3) {
    double p12[3];
    double p13[3];
    double n[3];

    for (int i = 0; i < 3; i++) {
        p12[i] = p2[i] - p1[i];
        p13[i] = p3[i] - p1[i];
    }

    isce::cuda::core::gpuLinAlg::cross(&p13[0], &p12[0], &n[0]);
    isce::cuda::core::gpuLinAlg::unitVec(&n[0], &nhat[0]);
}

__global__ void facet(float* out, size_t xmax, size_t ymax, float upsample_factor,
        isce::cuda::geometry::gpuDEMInterpolator dem_interp,
        isce::cuda::core::gpuEllipsoid ellps,
        isce::cuda::core::gpuOrbit orbit,
        isce::cuda::core::gpuLUT1d<double> dop,
        isce::cuda::product::gpuImageMode mode
        ) {

    const double RAD = M_PI / 180.;

    size_t xidx = threadIdx.x + blockIdx.x * blockDim.x;
    size_t yidx = threadIdx.y + blockIdx.y * blockDim.y;

    // Current latitude
    const double lat0 = dem_interp.yStart() + yidx * dem_interp.deltaY() / upsample_factor;
    const double lat1 = lat0 + dem_interp.deltaY() / upsample_factor;
    const double lat_mid = dem_interp.yStart() + (0.5 + yidx) * dem_interp.deltaY() / upsample_factor;

    using cartesian_t = double[3];
    cartesian_t xyz00, xyz01, xyz10, xyz11, xyz_mid,
                P00_01, P00_10, P10_01, P11_01, P11_10,
                lookXYZ, normalFacet1, normalFacet2;

    const double lon_mid = dem_interp.xStart() + dem_interp.deltaX() * (xidx + 0.5) / upsample_factor;

    double a, r;
    double inputLLH[3] = {lon_mid*RAD, lat_mid*RAD,
        dem_interp.interpolateXY(lon_mid, lat_mid)};

    isce::cuda::geometry::geo2rdr(&inputLLH[0], ellps, orbit, dop, mode,
            &a, &r, 1e-4, 100, 1e-4);

    const float azpix = (a - start) / pixazm;
    const float ranpix = (r - r0) / dr;

    // Establish bounds for bilinear weighting model
    const float x1 = std::floor(ranpix);
    const float x2 = x1 + 1.;
    const float y1 = std::floor(azpix);
    const float y2 = y1 + 1.;

    // Check to see if pixel lies in valid RDC range
    if (ranpix < 0.0 or x2 > xbound or azpix < 0.0 or y2 > ybound)
        return;

    // Current longitude
    const double lon0 = dem_interp.xStart() + dem_interp.deltaX() * xidx / upsample_factor;
    const double lon1 = lon0 + dem_interp.deltaX() / upsample_factor;

    // Set LLH vectors
    double llh00[] = {RAD*lon0, RAD*lat0,
        dem_interp.interpolateXY(lon0, lat0)};
    double llh01[] = {RAD*lon0, RAD*lat1,
        dem_interp.interpolateXY(lon0, lat1)};
    double llh10[] = {RAD*lon1, RAD*lat0,
        dem_interp.interpolateXY(lon1, lat0)};
    double llh11[] = {RAD*lon1, RAD*lat1,
        dem_interp.interpolateXY(lon1, lat1)};

    // Convert to XYZ
    ellps.lonLatToXyz(&llh00[0], xyz00);
    ellps.lonLatToXyz(&llh01[0], xyz01);
    ellps.lonLatToXyz(&llh10[0], xyz10);
    ellps.lonLatToXyz(&llh11[0], xyz11);

    // Compute normal vectors for each facet
    computePlaneNormal(normalFacet1, xyz00, xyz10, xyz01);
    computePlaneNormal(normalFacet2, xyz01, xyz10, xyz11);

    using isce::cuda::core::gpuLinAlg;

    // Compute vectors associated with facet sides
    gpuLinAlg::linComb(1., xyz00, -1., xyz01, P00_01);
    gpuLinAlg::linComb(1., xyz00, -1., xyz10, P00_10);
    gpuLinAlg::linComb(1., xyz10, -1., xyz01, P10_01);
    gpuLinAlg::linComb(1., xyz11, -1., xyz01, P11_01);
    gpuLinAlg::linComb(1., xyz11, -1., xyz10, P11_10);

    // Side lengths
    const double p00_01 = gpuLinAlg::norm(P00_01);
    const double p00_10 = gpuLinAlg::norm(P00_10);
    const double p10_01 = gpuLinAlg::norm(P10_01);
    const double p11_01 = gpuLinAlg::norm(P11_01);
    const double p11_10 = gpuLinAlg::norm(P11_10);

    // Semi-perimeters
    const float h1 = 0.5 * (p00_01 + p00_10 + p10_01);
    const float h2 = 0.5 * (p11_01 + p11_10 + p10_01);

    // Heron's formula to get area of facets in XYZ coordinates
    const float AP1 = std::sqrt(h1 * (h1 - p00_01) * (h1 - p00_10) * (h1 - p10_01));
    const float AP2 = std::sqrt(h2 * (h2 - p11_01) * (h2 - p11_10) * (h2 - p10_01));

    // Compute look angle from sensor to ground
    ellps.lonLatToXyz(inputLLH, xyz_mid);
    double xyz_plat[3];
    double vel[3];
    orbit.interpolateWGS84Orbit(a, &xyz_plat[0], &vel[0]);
    lookXYZ[0] = xyz_plat[0] - xyz_mid[0];
    lookXYZ[1] = xyz_plat[1] - xyz_mid[1];
    lookXYZ[2] = xyz_plat[2] - xyz_mid[2];

    double norm = gpuLinAlg::norm(lookXYZ);
    lookXYZ[0] /= norm;
    lookXYZ[1] /= norm;
    lookXYZ[2] /= norm;

    // Compute dot product between each facet and look vector
    const double cosIncFacet1 = gpuLinAlg::dot(lookXYZ, normalFacet1);
    const double cosIncFacet2 = gpuLinAlg::dot(lookXYZ, normalFacet2);

    // If facets are not illuminated by radar, skip
    if (cosIncFacet1 < 0. or cosIncFacet2 < 0.)
        return;

    // Compute projected area
    const float area = AP1 * cosIncFacet1 + AP2 * cosIncFacet2;

    // Get integer indices of bounds
    const int ix1 = static_cast<int>(x1);
    const int ix2 = static_cast<int>(x2);
    const int iy1 = static_cast<int>(y1);
    const int iy2 = static_cast<int>(y2);

    // Compute fractional weights from indices
    const float Wr = ranpix - x1;
    const float Wa = azpix - y1;
    const float Wrc = 1. - Wr;
    const float Wac = 1. - Wa;

    // Use bilinear weighting to distribute area
    atomicAdd(&out[mode.width() * iy1 + ix1], area * Wrc * Wac);
    atomicAdd(&out[mode.width() * iy1 + ix2], area * Wr  * Wac);
    atomicAdd(&out[mode.width() * iy2 + ix1], area * Wrc * Wa);
    atomicAdd(&out[mode.width() * iy2 + ix2], area * Wr  * Wa);
}

// Compute the flat earth incidence angle correction applied by UAVSAR processing
__global__ void flatearth(float* out,
        const isce::cuda::geometry::gpuDEMInterpolator flat_interp,
        const isce::cuda::core::gpuOrbit orbit,
        const isce::cuda::core::gpuEllipsoid ellps,
        const isce::cuda::product::gpuImageMode mode,
        float lookDir,
        float avg_hgt
        ) {
    size_t j = threadIdx.x + blockIdx.x * blockDim.x;
    size_t i = threadIdx.y + blockIdx.y * blockDim.y;

    if (j >= mode.width() or i >= mode.length())
        return;

    double xyz_plat[3];
    double vel[3];
    orbit.interpolateWGS84Orbit(start + i * pixazm, &xyz_plat[0], &vel[0]);

    // Slant range for current pixel
    const double slt_range = r0 + j * dr;

    // Get LLH and XYZ coordinates for this azimuth/range
    double targetLLH[3];
    double targetXYZ[3];
    targetLLH[2] = avg_hgt; // initialize first guess
    isce::cuda::geometry::rdr2geo(start + i * pixazm, slt_range, 0, orbit, ellps,
            flat_interp, targetLLH, mode.wavelength(), 1,
            1e-4, 20, 20);

    // Computation of ENU coordinates around ground target
    double satToGround[3];
    double enu[3];
    double enumat[9], xyz2enu[9];
    ellps.lonLatToXyz(targetLLH, targetXYZ);
    using isce::cuda::core::gpuLinAlg;
    gpuLinAlg::linComb(1., targetXYZ, -1., xyz_plat, satToGround);
    gpuLinAlg::enuBasis(targetLLH[1], targetLLH[0], enumat);
    gpuLinAlg::tranMat(enumat, xyz2enu);
    gpuLinAlg::matVec(xyz2enu, satToGround, enu);

    // Compute incidence angle components
    const double costheta = fabs(enu[2]) / gpuLinAlg::norm(enu);
    const double sintheta = sqrt(1. - costheta*costheta);

    out[mode.width() * i + j] *= sintheta;
}

double computeUpsamplingFactor(const isce::geometry::DEMInterpolator& dem_interp,
                               const isce::product::ImageMode& mode,
                               const isce::core::Ellipsoid& ellps) {
    // Create a projection object from the DEM interpolator
    isce::core::ProjectionBase * proj = isce::core::createProj(dem_interp.epsgCode());

    // Get middle XY coordinate in DEM coords, lat/lon, and ECEF XYZ
    isce::core::cartesian_t demXY{dem_interp.midX(), dem_interp.midY(), 0.}, llh;
    proj->inverse(demXY, llh);
    isce::core::cartesian_t xyz0;
    ellps.lonLatToXyz(llh, xyz0);

    // Repeat for middle coordinate + deltaX
    demXY[0] += dem_interp.deltaX();
    proj->inverse(demXY, llh);
    isce::core::cartesian_t xyz1;
    ellps.lonLatToXyz(llh, xyz1);

    // Repeat for middle coordinate + deltaX + deltaY
    demXY[1] += dem_interp.deltaY();
    proj->inverse(demXY, llh);
    isce::core::cartesian_t xyz2;
    ellps.lonLatToXyz(llh, xyz2);

    // Estimate width of DEM pixel
    isce::core::cartesian_t delta;
    isce::core::LinAlg::linComb(1., xyz1, -1., xyz0, delta);
    const double dx = isce::core::LinAlg::norm(delta);

    // Estimate length of DEM pixel
    isce::core::LinAlg::linComb(1., xyz2, -1., xyz1, delta);
    const double dy = isce::core::LinAlg::norm(delta);

    // Compute area of DEM pixel
    const double demArea = dx * dy;

    // Compute area of radar pixel (for now, just use spacing in range direction)
    const double radarArea = mode.rangePixelSpacing() * mode.rangePixelSpacing();

    // Upsampling factor is the ratio
    return std::sqrt(demArea / radarArea);
}

template<typename T>
T* deviceCopy(T& host_obj) {
    T* dev_obj;
    checkCudaErrors(hipMalloc(&dev_obj, sizeof(T)));
    checkCudaErrors(hipMemcpy(dev_obj, &host_obj, sizeof(T), hipMemcpyHostToDevice));
    return dev_obj;
}

template<typename T>
T* deviceCopy(T* host_obj) {
    T* dev_obj;
    checkCudaErrors(hipMalloc(&dev_obj, sizeof(T)));
    checkCudaErrors(hipMemcpy(dev_obj, host_obj, sizeof(T), hipMemcpyHostToDevice));
    return dev_obj;
}

namespace isce { namespace cuda {

    namespace core {
        using cartesian_t = double[3];
    };

    namespace geometry {

        void facetRTC(isce::product::Product& product,
                      isce::io::Raster& dem,
                      isce::io::Raster& out_raster) {

            isce::core::Ellipsoid ellps_h = product.metadata().identification().ellipsoid();
            isce::core::Orbit orbit_h(product.metadata().orbitPOE());
            isce::product::ImageMode mode_h = product.complexImagery().primaryMode();
            isce::geometry::Topo topo_h(product);
            topo_h.orbitMethod(isce::core::orbitInterpMethod::HERMITE_METHOD);

            // Initialize other ISCE objects
            isce::core::Peg peg;
            isce::core::Pegtrans ptm;
            ptm.radarToXYZ(ellps_h, peg);

            const double start_h = mode_h.startAzTime().secondsSinceEpoch();
            const double   end   = mode_h.  endAzTime().secondsSinceEpoch();
            const double pixazm_h = (end - start_h) / mode_h.length(); // azimuth difference per pixel
            const double r0_h = mode_h.startingRange();
            const double dr_h = mode_h.rangePixelSpacing();
            const float xbound_h = mode_h.width()  - 1.;
            const float ybound_h = mode_h.length() - 1.;
            checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(start),  &start_h, sizeof(start_h)));
            checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(pixazm), &pixazm_h, sizeof(pixazm_h)));
            checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(r0), &r0_h, sizeof(r0_h)));
            checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dr), &dr_h, sizeof(dr_h)));
            checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(xbound), &xbound_h, sizeof(xbound_h)));
            checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(ybound), &ybound_h, sizeof(ybound_h)));

            // Output raster
            auto out = std::make_unique<float[]>(mode_h.length() * mode_h.width());
            float* out_d;
            checkCudaErrors(hipMalloc(&out_d, mode_h.length() * mode_h.width() * sizeof(float)));

            // ------------------------------------------------------------------------
            // Main code: decompose DEM into facets, compute RDC coordinates
            // ------------------------------------------------------------------------

            // Create CPU-only  objects
            isce::geometry::DEMInterpolator dem_interp_h(0, isce::core::dataInterpMethod::BIQUINTIC_METHOD);
            topo_h.computeDEMBounds(dem, dem_interp_h, 0, mode_h.length()); // determine DEM bounds

            const float upsample_factor = computeUpsamplingFactor(dem_interp_h, mode_h, ellps_h);

            float max_hgt, avg_hgt;
            pyre::journal::info_t info("gpuRTC");
            dem_interp_h.computeHeightStats(max_hgt, avg_hgt, info);
            isce::cuda::geometry::gpuDEMInterpolator flat_interp(avg_hgt);

            // Create hostside device objects
            isce::cuda::geometry::gpuDEMInterpolator dem_interp(dem_interp_h);
            isce::cuda::core::gpuEllipsoid ellps(ellps_h);
            isce::cuda::core::gpuOrbit orbit(orbit_h);
            isce::cuda::core::gpuLUT1d<double> dop(product.metadata().instrument().skewDoppler());
            isce::cuda::product::gpuImageMode mode(mode_h);

            const size_t xmax = dem_interp.width()  * upsample_factor;
            const size_t ymax = dem_interp.length() * upsample_factor;

            dem_interp.initProjInterp();

#define BLOCK_X 16
#define BLOCK_Y 16
            static_assert(BLOCK_X * BLOCK_Y <= 256,
                    "RTC block dim too large for resources available on GPU");

            {
                dim3 block(BLOCK_X, BLOCK_Y);
                dim3 grid(xmax / BLOCK_X + 1,
                          ymax / BLOCK_Y + 1);
                facet<<<grid, block>>>(out_d, xmax, ymax, upsample_factor,
                                       dem_interp, ellps, orbit, dop, mode);
                checkCudaErrors(hipPeekAtLastError());
                checkCudaErrors(hipDeviceSynchronize());
            }

            {
                dim3 block(BLOCK_X, BLOCK_Y);
                dim3 grid(mode.width() / BLOCK_X + 1,
                          mode.width() / BLOCK_Y + 1);
                flatearth<<<grid, block>>>(out_d, flat_interp, orbit, ellps, mode,
                        product.metadata().identification().lookDirection(), avg_hgt);
                checkCudaErrors(hipPeekAtLastError());
                checkCudaErrors(hipDeviceSynchronize());
            }

            checkCudaErrors(hipMemcpy(&out[0], out_d, mode.width() * mode.length() * sizeof(float),
                                       hipMemcpyDeviceToHost));
            out_raster.setBlock(&out[0], 0, 0, mode.width(), mode.length());
        }
    };

}; };
