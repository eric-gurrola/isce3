#include "hip/hip_runtime.h"
//-*- C++ -*-
//-*- coding: utf-8 -*-
//
// Author: Joshua Cohen, Bryan V. Riel, Liang Yu
// Copyright 2017-2019
//

#include <algorithm>
#include <iostream>
#include <chrono>
#include <cmath>
#include <thrust/transform_reduce.h>
#include <thrust/advance.h>

// isce::core
#include <isce/core/Constants.h>

// isce::cuda::core
#include <isce/cuda/core/gpuPoly2d.h>
#include <isce/cuda/core/gpuLUT1d.h>
#include <isce/cuda/core/Stream.h>

#include <isce/cuda/io/DataStream.h>

#include "ResampSlc.h"
#include "gpuResampSlc.h"

using isce::io::Raster;

using isce::cuda::core::gpuPoly2d;
using isce::cuda::core::gpuInterpolator;
using isce::cuda::core::gpuLUT1d;
using isce::cuda::core::gpuSinc2dInterpolator;

// thrust::host_vector whose data buffer uses page-locked memory
template<typename T>
using pinned_host_vector = thrust::host_vector<T,
        thrust::system::cuda::experimental::pinned_allocator<T>>;

//using isce::cuda::core::Stream;

//using isce::cuda::io::DataStream;

#define THRD_PER_BLOCK 512// Number of threads per block (should always %32==0)
__global__
void removeCarrier(thrust::complex<float> *tile,
        int firstImageRow,
        int outWidth,
        int outLength,
        const gpuPoly2d rgCarrier,
        const gpuPoly2d azCarrier) {

    int iTile = blockDim.x * blockIdx.x + threadIdx.x;

    if (iTile < outWidth*outLength) {
        double phase = fmod(rgCarrier.eval(blockIdx.x+firstImageRow, threadIdx.x)
                + azCarrier.eval(blockIdx.x+firstImageRow, threadIdx.x), 2.0*M_PI);
        tile[iTile] *= thrust::complex<float>(cos(phase), -sin(phase));
    }
}

__global__
void transformTile(const thrust::complex<float> *tile,
                   thrust::complex<float> *chip,
                   thrust::complex<float> *imgOut,
                   const float *rgOffTile,
                   const float *azOffTile,
                   const gpuPoly2d rgCarrier,
                   const gpuPoly2d azCarrier,
                   const gpuLUT1d<double> dopplerLUT,
                   gpuSinc2dInterpolator<thrust::complex<float>> interp,
                   bool flatten,
                   int outWidth,
                   int outLength,
                   int inWidth,
                   int inLength,
                   double startingRange,
                   double rangePixelSpacing,
                   double prf,
                   double wavelength,
                   double refStartingRange,
                   double refRangePixelSpacing,
                   double refWavelength,
                   int chipSize,
                   int rowOffset, 
                   int rowStart) {

    int iTileOut = blockDim.x * blockIdx.x + threadIdx.x;
    int iChip = iTileOut * chipSize * chipSize;                                          
    int chipHalf = chipSize/2;

    if (iTileOut < outWidth*outLength) {
        int i = iTileOut / outWidth;
        int j = iTileOut % outWidth;
        imgOut[iTileOut] = thrust::complex<float>(0., 0.);

        // Unpack offsets
        const float azOff = azOffTile[iTileOut];
        const float rgOff = rgOffTile[iTileOut];

        // Break into fractional and integer parts
        const int intAz = __float2int_rd(i + azOff + rowStart);
        const int intRg = __float2int_rd(j + rgOff);
        const double fracAz = i + azOff - intAz + rowStart;
        const double fracRg = j + rgOff - intRg;
       
        // Check bounds again. Use rowOffset to account tiles where rowStart != firstRowImage
        bool intAzInBounds = !((intAz+rowOffset < chipHalf) || (intAz >= (inLength - chipHalf)));
        bool intRgInBounds = !((intRg < chipHalf) || (intRg >= (inWidth - chipHalf)));

        if (intAzInBounds && intRgInBounds) {
            // evaluate Doppler polynomial
            const double rng = startingRange + j * rangePixelSpacing;
            const double dop = dopplerLUT.eval(rng) * 2 * M_PI / prf;

            // Doppler to be added back. Simultaneously evaluate carrier that needs to
            // be added back after interpolation
            double phase = (dop * fracAz) 
                + rgCarrier.eval(i + azOff, j + rgOff) 
                + azCarrier.eval(i + azOff, j + rgOff);

            // Flatten the carrier phase if requested
            if (flatten) {
                phase += ((4. * (M_PI / wavelength)) * 
                    ((startingRange - refStartingRange) 
                    + (j * (rangePixelSpacing - refRangePixelSpacing)) 
                    + (rgOff * rangePixelSpacing))) + ((4.0 * M_PI 
                    * (refStartingRange + (j * refRangePixelSpacing))) 
                    * ((1.0 / refWavelength) - (1.0 / wavelength)));
            }
            
            // Modulate by 2*PI
            phase = fmod(phase, 2.0*M_PI);
            
            // Read data chip without the carrier phases
            for (int ii = 0; ii < chipSize; ++ii) {
                // Row to read from
                const int chipRow = intAz + ii - chipHalf + rowOffset - rowStart;
                // Carrier phase
                const double phase = dop * (ii - 4.0);
                const thrust::complex<float> cval(cos(phase), -sin(phase));
                // Set the data values after removing doppler in azimuth
                for (int jj = 0; jj < chipSize; ++jj) {
                    // Column to read from
                    const int chipCol = intRg + jj - chipHalf;
                    chip[iChip + ii*chipSize+jj] = tile[chipRow*outWidth+chipCol] * cval;
                }
            }

            // Interpolate chip
            const thrust::complex<float> cval = interp.interpolate(
                chipHalf + fracRg, chipHalf + fracAz, &chip[iChip], chipSize, chipSize
            );

            // Add doppler to interpolated value and save
            imgOut[iTileOut] = cval * thrust::complex<float>(cos(phase), sin(phase));
        }
    }
}


// Alternative generic resamp entry point: use filenames to internally create rasters
void isce::cuda::image::ResampSlc::
resamp(const std::string & inputFilename,          // filename of input SLC
       const std::string & outputFilename,         // filename of output resampled SLC
       const std::string & rgOffsetFilename,       // filename of range offsets
       const std::string & azOffsetFilename,       // filename of azimuth offsets
       int inputBand, bool flatten, bool isComplex, int rowBuffer,
       int chipSize) {

    // Make input rasters
    Raster inputSlc(inputFilename, GA_ReadOnly);
    Raster rgOffsetRaster(rgOffsetFilename, GA_ReadOnly);
    Raster azOffsetRaster(azOffsetFilename, GA_ReadOnly);

    // Make output raster; geometry defined by offset rasters
    const int outLength = rgOffsetRaster.length();
    const int outWidth = rgOffsetRaster.width();
    Raster outputSlc(outputFilename, outWidth, outLength, 1, GDT_CFloat32, "ISCE");

    // Call generic resamp
    resamp(inputSlc, outputSlc, rgOffsetRaster, azOffsetRaster, inputBand, flatten,
           isComplex, rowBuffer, chipSize);
}

// Generic resamp entry point from externally created rasters
void isce::cuda::image::ResampSlc::
resamp(isce::io::Raster & inputSlc, isce::io::Raster & outputSlc,
       isce::io::Raster & rgOffsetRaster, isce::io::Raster & azOffsetRaster,
       int inputBand, bool flatten, bool isComplex, int rowBuffer,
       int chipSize) {

    // Check if data are not complex
    if (!isComplex) {
        std::cout << "Real data interpolation not implemented yet.\n";
        return;
    }
    // Set the band number for input SLC
    _inputBand = inputBand;
    // Cache width of SLC image
    const int inLength = inputSlc.length();
    const int inWidth = inputSlc.width();
    // Cache output length and width from offset images
    const int outLength = rgOffsetRaster.length();
    const int outWidth = rgOffsetRaster.width();

    // Check if reference data is available
    if (!this->haveRefData()) {
        flatten = false;
    }

    // initialize interpolator
    isce::cuda::core::gpuSinc2dInterpolator<thrust::complex<float>> interp(chipSize-1, isce::core::SINC_SUB);

    // Determine number of tiles needed to process image
    const int nTiles = isce::image::_computeNumberOfTiles(outLength, _linesPerTile);
    std::cout << 
        "GPU resampling using " << nTiles << " tiles of " << _linesPerTile 
        << " lines per tile\n";
    // Start timer
    auto timerStart = std::chrono::steady_clock::now();

    // For each full tile of _linesPerTile lines...
    for (int tileCount = 0; tileCount < nTiles; tileCount++) {

        int rowStart = tileCount * _linesPerTile;
        int rowEnd(rowStart + _linesPerTile);
        if (tileCount == (nTiles - 1)) {
            rowEnd = outLength;
        }
        int outLength = rowEnd - rowStart;
        int nOutPixels = outWidth * outLength;

        // initialize range offsets
        thrust::device_vector<float> d_rgOffsets(nOutPixels);
        pinned_host_vector<float> h_rgOffsets(nOutPixels);
        rgOffsetRaster.getBlock(h_rgOffsets.data(), 0, rowStart, outWidth, outLength);
        isce::cuda::core::Stream streamRgOffset;
        checkCudaErrors( hipMemcpyAsync(d_rgOffsets.data().get(), h_rgOffsets.data(),
                    nOutPixels*sizeof(float), hipMemcpyHostToDevice, streamRgOffset.get()) );

        /*
        isce::cuda::core::Stream streamRgOffset;
        isce::cuda::io::RasterDataStream datastreamRgOffset(&rgOffsetRaster, streamRgOffset);
        datastreamRgOffset.load(d_rgOffsets.data().get(), 0, rowStart, outWidth, outLength);
        */

        // initialize azimuth offsets
        thrust::device_vector<float> d_azOffsets(nOutPixels);
        pinned_host_vector<float> h_azOffsets(nOutPixels);
        azOffsetRaster.getBlock(h_azOffsets.data(), 0, rowStart, outWidth, outLength);
        isce::cuda::core::Stream streamAzOffset;
        checkCudaErrors( hipMemcpyAsync(d_azOffsets.data().get(), h_azOffsets.data(),
                    nOutPixels*sizeof(float), hipMemcpyHostToDevice, streamRgOffset.get()) );

        /*
        isce::cuda::core::Stream streamAzOffset;
        isce::cuda::io::RasterDataStream datastreamAzOffset(&azOffsetRaster, streamAzOffset);
        datastreamAzOffset.load(d_azOffsets.data().get(), 0, rowStart, outWidth, outLength);
        */

        // prepare SLC
        // Compute minimum row index needed from input image
        int firstImageRow(outLength - 1);
        bool haveOffsets = false;
        int chipHalf = chipSize/2;
        for (int i = 0; i < std::min(rowBuffer, outLength); ++i) {
            for (int j = 0; j < outWidth; ++j) {
                // Get azimuth offset for pixel
                const double azOff = d_azOffsets[i,j];
                // Skip null values
                if (azOff < -5.0e5 || std::isnan(azOff)) {
                    continue;
                } else {
                    haveOffsets = true;
                }
                // Calculate corresponding minimum line index of input image
                const int imageLine = static_cast<int>(i + azOff + rowStart - chipHalf);
                // Update minimum row index
                firstImageRow = std::min(firstImageRow, imageLine);
            }
        }
        if (haveOffsets) {
            firstImageRow = std::max(firstImageRow, 0);
        } else {
            firstImageRow = 0;
        }

        // Compute maximum row index needed from input image
        int lastImageRow(0);
        haveOffsets = false;
        for (int i = std::max(outLength - rowBuffer, 0); i < outLength; ++i) {
            for (int j = 0; j < outWidth; ++j) {
                // Get azimuth offset for pixel
                const double azOff = d_azOffsets[i,j];
                // Skip null values 
                if (azOff < -5.0e5 || std::isnan(azOff)) {
                    continue;
                } else {
                    haveOffsets = true;
                }
                // Calculate corresponding minimum line index of input image
                const int imageLine = static_cast<int>(i + azOff + rowStart + chipHalf);
                // Update maximum row index
                lastImageRow = std::max(lastImageRow, imageLine);
            }
        }
        if (haveOffsets) {
            lastImageRow = std::min(lastImageRow + 1, inLength);
        } else {
            lastImageRow = inLength;
        }

        // Get corresponding image indices
        // replace below with stream init
        std::cout << "Reading in image data for tile " << tileCount << std::endl;
        int nInPixels = inWidth * (lastImageRow-firstImageRow);
        thrust::device_vector<thrust::complex<float>> d_slc(nInPixels);
        pinned_host_vector<thrust::complex<float>> h_slc(nInPixels);
        inputSlc.getBlock(h_slc.data(), 0, firstImageRow, inWidth, lastImageRow-firstImageRow, _inputBand);
        isce::cuda::core::Stream streamSlc;
        checkCudaErrors( hipMemcpyAsync(d_slc.data().get(), h_slc.data(),
                    nInPixels*sizeof(thrust::complex<float>), hipMemcpyHostToDevice, streamSlc.get()) );

        /*
        isce::cuda::core::Stream streamSlc;
        isce::cuda::io::RasterDataStream datastreamSlc(&inputSlc, streamSlc);
        inputSlc.getBlock(d_slc.data().get(), 0, firstImageRow, inWidth, lastImageRow-firstImageRow, _inputBand);
        */

        thrust::device_vector<thrust::complex<float>> d_chip(nOutPixels * chipSize * chipSize); // make contiguous 2D
        thrust::device_vector<thrust::complex<float>> d_imgOut(nOutPixels); // tie in with RasterDataStream
        gpuPoly2d d_rgCarrier(_rgCarrier);
        gpuPoly2d d_azCarrier(_azCarrier);
        gpuLUT1d<double> d_dopplerLUT(_dopplerLUT);

        // move phase removal computation into device
        // determine block layout
        dim3 block(THRD_PER_BLOCK);
        dim3 grid((nInPixels+(THRD_PER_BLOCK-1))/THRD_PER_BLOCK);
        removeCarrier<<<grid, block>>>(d_slc.data().get(), firstImageRow, outWidth, outLength, _rgCarrier, _azCarrier);
        // TODO synchronize before resampling

        // Perform interpolation
        // determine block layout
        grid = (nOutPixels+(THRD_PER_BLOCK-1)) / THRD_PER_BLOCK;
        int rowOffset = rowStart-firstImageRow;
        std::cout << "Interpolating tile " << tileCount << std::endl;
        transformTile<<<grid, block>>>(d_slc.data().get(),
                d_chip.data().get(),
                d_imgOut.data().get(),
                d_rgOffsets.data().get(), d_azOffsets.data().get(),
                d_rgCarrier, d_azCarrier, d_dopplerLUT,
                interp, flatten,
                outWidth, outLength,
                inWidth, inLength,
                this->startingRange(), this->rangePixelSpacing(),
                this->prf(), this->wavelength(),
                this->refStartingRange(), this->refRangePixelSpacing(),
                this->refWavelength(), chipSize,
                rowOffset,// needed to keep az in bounds in subtiles
                rowStart); // needed to match az components on CPU

    }

    // Print out timing information and reset
    auto timerEnd = std::chrono::steady_clock::now();
    const double elapsed = 1.0e-3 * std::chrono::duration_cast<std::chrono::milliseconds>(
        timerEnd - timerStart).count();
    std::cout << "Elapsed processing time: " << elapsed << " sec" << "\n";
}

// end of file
