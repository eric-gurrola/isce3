#include "hip/hip_runtime.h"
//-*- coding: utf-8 -*-
//
// Author: Liang Yu
// Copyright: 2018
#include <math.h>

// isce::core
#include "isce/core/Constants.h"
#include "isce/core/Poly2d.h"

// isce::cuda::core
#include "isce/cuda/core/gpuPoly2d.h"
#include "isce/cuda/core/gpuLUT1d.h"

// isce::cuda::image
#include "gpuResampSlc.h"
#include "gpuImageMode.h"

#include "isce/cuda/hip/hip_runtime_api.h"
#include <fstream>
#include <string>
using isce::cuda::core::gpuComplex;
using isce::cuda::core::gpuPoly2d;
using isce::cuda::core::gpuInterpolator;
using isce::cuda::core::gpuLUT1d;
using isce::cuda::core::gpuSinc2dInterpolator;
using isce::cuda::image::gpuImageMode;

#define THRD_PER_BLOCK 512// Number of threads per block (should always %32==0)

__global__
void transformTile(const gpuComplex<float> *tile,
                   gpuComplex<float> *chip,
                   gpuComplex<float> *imgOut,
                   const float *rgOffTile,
                   const float *azOffTile,
                   const gpuPoly2d rgCarrier,
                   const gpuPoly2d azCarrier,
                   const gpuLUT1d<double> dopplerLUT,
                   gpuImageMode mode,       // image mode for image to be resampled
                   gpuImageMode refMode,    // image mode for reference master image
                   gpuSinc2dInterpolator<gpuComplex<float>> interp,
                   bool flatten,
                   int outWidth,
                   int outLength,
                   int inWidth,
                   int inLength,
                   int chipSize,
                   int rowOffset, 
                   int rowStart) {

    int iTileOut = blockDim.x * blockIdx.x + threadIdx.x;
    int iChip = iTileOut * chipSize * chipSize;                                          
    int chipHalf = chipSize/2;

    if (iTileOut < outWidth*outLength) {
        int i = iTileOut / outWidth;
        int j = iTileOut % outWidth;
        imgOut[iTileOut] = gpuComplex<float>(0., 0.);

        // Unpack offsets
        const float azOff = azOffTile[iTileOut];
        const float rgOff = rgOffTile[iTileOut];

        // Break into fractional and integer parts
        const int intAz = __float2int_rd(i + azOff + rowStart);
        const int intRg = __float2int_rd(j + rgOff);
        const double fracAz = i + azOff - intAz + rowStart;
        const double fracRg = j + rgOff - intRg;
       
        // Check bounds again. Use rowOffset to account tiles where tile.rowStart != tile.firstRowImage
        bool intAzInBounds = !((intAz+rowOffset < chipHalf) || (intAz >= (inLength - chipHalf)));
        bool intRgInBounds = !((intRg < chipHalf) || (intRg >= (inWidth - chipHalf)));

        if (intAzInBounds && intRgInBounds) {
            // evaluate Doppler polynomial
            const double rng = mode.startingRange + j * mode.rangePixelSpacing;
            const double dop = dopplerLUT.eval(rng) * 2 * M_PI / mode.prf;

            // Doppler to be added back. Simultaneously evaluate carrier that needs to
            // be added back after interpolation
            double phase = (dop * fracAz) 
                + rgCarrier.eval(i + azOff, j + rgOff) 
                + azCarrier.eval(i + azOff, j + rgOff);

            // Flatten the carrier phase if requested
            if (flatten && refMode.isRefMode) {
                phase += ((4. * (M_PI / mode.wavelength)) * 
                    ((mode.startingRange - refMode.startingRange) 
                    + (j * (mode.rangePixelSpacing - refMode.rangePixelSpacing)) 
                    + (rgOff * mode.rangePixelSpacing))) + ((4.0 * M_PI 
                    * (refMode.startingRange + (j * refMode.rangePixelSpacing))) 
                    * ((1.0 / refMode.wavelength) - (1.0 / mode.wavelength)));
            }
            
            // Modulate by 2*PI
            phase = fmod(phase, 2.0*M_PI);
            
            // Read data chip without the carrier phases
            for (int ii = 0; ii < chipSize; ++ii) {
                // Row to read from
                const int chipRow = intAz + ii - chipHalf + rowOffset - rowStart;
                // Carrier phase
                const double phase = dop * (ii - 4.0);
                const gpuComplex<float> cval(cos(phase), -sin(phase));
                // Set the data values after removing doppler in azimuth
                for (int jj = 0; jj < chipSize; ++jj) {
                    // Column to read from
                    const int chipCol = intRg + jj - chipHalf;
                    chip[iChip + ii*chipSize+jj] = tile[chipRow*outWidth+chipCol] * cval;
                }
            }

            // Interpolate chip
            const gpuComplex<float> cval = interp.interpolate(
                chipHalf + fracRg, chipHalf + fracAz, &chip[iChip], chipSize, chipSize
            );

            // Add doppler to interpolated value and save
            imgOut[iTileOut] = cval * gpuComplex<float>(cos(phase), sin(phase));
        }
    }
}


// Interpolate tile to perform transformation
void isce::cuda::image::
gpuTransformTile(isce::image::Tile<std::complex<float>> & tile,
               isce::io::Raster & outputSlc,
               isce::image::Tile<float> & rgOffTile,
               isce::image::Tile<float> & azOffTile,
               const isce::core::Poly2d & rgCarrier,
               const isce::core::Poly2d & azCarrier,
               const isce::core::LUT1d<double> & dopplerLUT,
               isce::product::ImageMode mode,       // image mode for image to be resampled
               isce::product::ImageMode refMode,    // image mode for reference master image
               bool haveRefMode,
               isce::cuda::core::gpuSinc2dInterpolator<gpuComplex<float>> interp,
               int inWidth, int inLength, bool flatten, int chipSize) {

    // Cache geometry values
    const int outWidth = azOffTile.width();
    const int outLength = azOffTile.length();

    // Allocate valarray for output image block
    std::valarray<std::complex<float>> imgOut(outLength * outWidth);
    // Initialize to zeros
    imgOut = std::complex<float>(0.0, 0.0);

    // declare equivalent objects in device memory
    gpuComplex<float> *d_tile;
    gpuComplex<float> *d_chip;
    gpuComplex<float> *d_imgOut;
    float *d_rgOffTile, *d_azOffTile;
    gpuPoly2d d_rgCarrier(rgCarrier);
    gpuPoly2d d_azCarrier(azCarrier);
    gpuImageMode d_mode(mode);
    gpuImageMode d_refMode;             // empty by default
    if (haveRefMode)
        gpuImageMode d_mode(refMode);   // populate from CPU version if provided
    gpuLUT1d<double> d_dopplerLUT(dopplerLUT);

    // determine sizes
    size_t nInPixels = (tile.lastImageRow() - tile.firstImageRow() + 1) * outWidth;
    size_t nOutPixels = imgOut.size();
    size_t nOutBytes = nOutPixels * sizeof(gpuComplex<float>);
    size_t nChipBytes = nOutBytes * chipSize * chipSize;

    // allocate equivalent objects in device memory
    checkCudaErrors(hipMalloc(&d_tile, nInPixels*sizeof(gpuComplex<float>)));
    checkCudaErrors(hipMalloc(&d_chip, nChipBytes));
    checkCudaErrors(hipMalloc(&d_imgOut, nOutBytes));
    checkCudaErrors(hipMalloc(&d_azOffTile, nOutPixels*sizeof(float)));
    checkCudaErrors(hipMalloc(&d_rgOffTile, nOutPixels*sizeof(float)));

    // copy objects to device memory
    checkCudaErrors(hipMemcpy(d_tile, &tile[0], nInPixels*sizeof(gpuComplex<float>), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_azOffTile, &azOffTile[0], nOutPixels*sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_rgOffTile, &rgOffTile[0], nOutPixels*sizeof(float), hipMemcpyHostToDevice));

    // determine block layout
    dim3 block(THRD_PER_BLOCK);
    dim3 grid((nOutPixels+(THRD_PER_BLOCK-1))/THRD_PER_BLOCK);

    // global call to transform
    transformTile<<<grid, block>>>(d_tile, 
                                   d_chip,
                                   d_imgOut, 
                                   d_rgOffTile, 
                                   d_azOffTile, 
                                   d_rgCarrier, 
                                   d_azCarrier, 
                                   d_dopplerLUT, 
                                   d_mode, 
                                   d_refMode,
                                   interp,
                                   flatten,
                                   outWidth,
                                   outLength,
                                   inWidth,
                                   inLength,
                                   chipSize,
                                   tile.rowStart()-tile.firstImageRow(),// needed to keep az in bounds in subtiles
                                   tile.rowStart());                    // needed to match az components on CPU

    // Check for any kernel errors
    checkCudaErrors(hipPeekAtLastError());

    // copy to host memory
    checkCudaErrors(hipMemcpy(&imgOut[0], d_imgOut, nOutBytes, hipMemcpyDeviceToHost));

    // deallocate to device memory
    checkCudaErrors(hipFree(d_tile));
    checkCudaErrors(hipFree(d_chip));
    checkCudaErrors(hipFree(d_imgOut));
    checkCudaErrors(hipFree(d_azOffTile));
    checkCudaErrors(hipFree(d_rgOffTile));
    
    // Write block of data
    outputSlc.setBlock(imgOut, 0, tile.rowStart(), outWidth, outLength);
}
