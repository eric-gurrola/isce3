#include "hip/hip_runtime.h"
//
// Author: Liang Yu
// Copyright 2018
//

#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "gpuInterpolator.h"
#include "../hip/hip_runtime_api.h"

#define MAX_ORDER 20

using isce::cuda::core::gpuInterpolator;
using isce::cuda::core::gpuSpline2dInterpolator;


/*
   each derived class needs it's own wrapper_d, gpuInterpolator_g...
*/

template <class U>
__device__ void wrapper_d(gpuSpline2dInterpolator<U> interp, double x, double y, const U *z, U *value, size_t nx, size_t ny=0) {
    /*
     *  device side wrapper used to get map interfaces of actual device function to global test function
     */
    *value = interp.interpolate(x, y, z, nx, ny);
}


template <class U>
__global__ void gpuInterpolator_g(gpuSpline2dInterpolator<U> interp, double *x, double *y, const U *z, U *value, size_t nx, size_t ny=0) {
    /*
     *  GPU kernel to test interpolate() on the device for consistency.
     */
    int i = threadIdx.x;
    wrapper_d(interp, x[i], y[i], z, &value[i], nx, ny);
}


template <class U>
__host__ void gpuSpline2dInterpolator<U>::interpolate_h(const Matrix<double>& truth, Matrix<U>& m, double start, double delta, U* h_z) {
    /*
     *  CPU-side function to call the corresponding GPU function on a single thread for consistency checking
     */

    // allocate host side memory
    size_t size_input_pts = truth.length() * sizeof(double);
    size_t size_output_pts = truth.length() * sizeof(U);
    double *h_x = (double *)malloc(size_input_pts);
    double *h_y = (double *)malloc(size_input_pts);
    size_t nx = m.width();
    size_t ny = m.length();

    // assign host side inputs
    for (size_t i = 0; i < truth.length(); ++i) {
        h_x[i] = (truth(i,0) - start) / delta;
        h_y[i] = (truth(i,1) - start) / delta;
    }

    // allocate devie side memory
    double *d_x;
    checkCudaErrors(hipMalloc((void**)&d_x, size_input_pts));
    double *d_y;
    checkCudaErrors(hipMalloc((void**)&d_y, size_input_pts));
    U *d_z;
    checkCudaErrors(hipMalloc((void**)&d_z, size_output_pts));
    U *d_m;
    checkCudaErrors(hipMalloc((U**)&d_m, m.length()*m.width()*sizeof(U)));

    // copy input data
    checkCudaErrors(hipMemcpy(d_x, h_x, size_input_pts, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_y, h_y, size_input_pts, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_m, &m.data()[0], m.length()*m.width()*sizeof(U), hipMemcpyHostToDevice)); 

    // launch!
    int n_threads = truth.length();
    gpuInterpolator_g<U><<<1, n_threads>>>(*this, d_x, d_y, d_m, d_z, nx, ny);
    
    // copy device results to host
    checkCudaErrors(hipMemcpy(h_z, d_z, size_output_pts, hipMemcpyDeviceToHost));

    // free memory
    checkCudaErrors(hipFree(d_x));
    checkCudaErrors(hipFree(d_y));
    checkCudaErrors(hipFree(d_z));
    checkCudaErrors(hipFree(d_m));
}


__device__ void initSpline_d(double *Y, int n, double *R, double *Q) {
    Q[0] = 0.0;
    R[0] = 0.0;
    for (int i = 1; i < n - 1; ++i) {
        const double p = 1.0 / (0.5 * Q[i-1] + 2.0);
        Q[i] = -0.5 * p;
        R[i] = (3 * (Y[i+1] - 2*Y[i] + Y[i-1]) - 0.5*R[i-1]) * p;
    }
    R[n-1] = 0.0;
    for (int i = (n - 2); i > 0; --i)
        R[i] = Q[i] * R[i+1] + R[i];
}


__device__ double spline_d(double x, double *Y, int n, double * R) {
    if (x < 1.0) {
        return Y[0] + (x - 1.0) * (Y[1] - Y[0] - (R[1] / 6.0));
    } else if (x > n) {
        return Y[n-1] + ((x - n) * (Y[n-1] - Y[n-2] + (R[n-2] / 6.)));
    } else {
        int j = int(floor(x));
        auto xx = x - j;
        auto t0 = Y[j] - Y[j-1] - (R[j-1] / 3.0) - (R[j] / 6.0);
        auto t1 = xx * ((R[j-1] / 2.0) + (xx * ((R[j] - R[j-1]) / 6)));
        return Y[j-1] + (xx * (t0 + t1));
    }
}


template <class U>
__device__ U gpuSpline2dInterpolator<U>::interpolate(double x, double y, const U *z, size_t nx, size_t ny=0) {
    // Get coordinates of start of spline window
    int i0, j0;
    
    if ((_order % 2) != 0) {
        i0 = y - 0.5;
        j0 = x - 0.5;
    } else {
        i0 = y;
        j0 = x;
    }
    i0 = i0 - (_order / 2) + 1;
    j0 = j0 - (_order / 2) + 1;

    double A[MAX_ORDER]={0}, R[MAX_ORDER]={0}, Q[MAX_ORDER]={0}, HC[MAX_ORDER]={0};

    for (int i = 0; i < _order; ++i) {
        const int indi = min(max(i0 + i, 0), int(ny) - 2);
        for (int j = 0; j < _order; ++j) {
            const int indj = min(max(j0 + j, 0), int(nx) - 2);
            A[j] = z[(indi+1)*nx +indj+1];
        }
        initSpline_d(A, _order, R, Q);
        HC[i] = spline_d(x - j0, A, _order, R);
    }

    initSpline_d(HC, _order, R, Q);
    U spline_out = (U)(spline_d(y - i0, HC, _order, R));

    return spline_out;
}


/*
 each template parameter needs it's own declaration here
 */
template class gpuSpline2dInterpolator<double>;
template class gpuSpline2dInterpolator<float>;

template __global__ void
gpuInterpolator_g<double>(gpuSpline2dInterpolator<double> interp, double *x, double *y,
                          const double *z, double *value, size_t nx, size_t ny);
