#include "hip/hip_runtime.h"
//
// Author: Liang Yu
// Copyright 2018
//
// NOTE: gpuOrbit used as template

#include <hip/hip_runtime.h>
#include <vector>
#include "isce/core/Constants.h"
#include "gpuPoly2d.h"
#include <stdio.h>

using isce::cuda::core::gpuPoly2d;
using isce::core::Poly2d;
using std::vector;


// Advanced "copy" constructor to handle deep-copying of Poly2d data (only callable by host). Owner 
// member variable indicates that only the host-side copy of the gpuPoly2d can handle freeing the 
// memory (device-side copy constructor for gpuPoly2d sets owner to false)
__host__ gpuPoly2d::gpuPoly2d(const Poly2d &poly) :
    rangeOrder(poly.rangeOrder), 
    azimuthOrder(poly.azimuthOrder), 
    rangeMean(poly.rangeMean), 
    azimuthMean(poly.azimuthMean),
    rangeNorm(poly.rangeNorm), 
    azimuthNorm(poly.azimuthNorm),
    owner(true)
{
    
    const int n_coeffs = poly.coeffs.size();

    // Malloc device-side memory (this API is host-side only)
    hipMalloc(&coeffs, n_coeffs*sizeof(double));

    // Copy OrPoly2d data to device-side memory and keep device pointer in gpuOrPoly2d object. Device-side 
    // copy constructor simply shallow-copies the device pointers when called
    hipMemcpy(coeffs, poly.coeffs.data(), n_coeffs*sizeof(double), hipMemcpyHostToDevice);
}


// Both the host-side and device-side copies of the gpuPoly2d will call the destructor, so we have to 
// implement a way of having an arbitrary copy on host OR device determine when to free the memory 
// (since only the original host-side copy should free)
gpuPoly2d::~gpuPoly2d() {
    if (owner) {
        hipFree(coeffs);
    }
}

__device__ double gpuPoly2d::eval(double azi, double rng) const {

    double xval = (rng - rangeMean) / rangeNorm;
    double yval = (azi - azimuthMean) / azimuthNorm;

    double scalex;
    double scaley = 1.;
    double val = 0.;
    for (int i=0; i<=azimuthOrder; i++,scaley*=yval) {
        scalex = 1.;
        for (int j=0; j<=rangeOrder; j++,scalex*=xval) {
            val += scalex * scaley * coeffs[IDX1D(i,j,rangeOrder+1)];
        }
    }

    return val;
}

__global__ void eval_d(gpuPoly2d p, double azi, double rng, double *val)
{
    *val = p.eval(azi, rng);
}

__host__ double gpuPoly2d::eval_h(double azi, double rng)
{
    double *val_d;
    double val_h;
    // use unified memory?
    hipMalloc((double**)&val_d, sizeof(double));
    dim3 grid(1), block(1);
    eval_d<<<grid,block>>>(*this, azi, rng, val_d);
    hipMemcpy(&val_h, val_d, sizeof(double), hipMemcpyDeviceToHost);
    hipFree(val_d);
    return val_h;
}

