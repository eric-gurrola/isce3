#include "hip/hip_runtime.h"
//
// Author: Joshua Cohen, Liang Yu
// Copyright 2017-2018
//

#include <cmath>
#include <hip/hip_runtime.h>
#include <vector>
#include "gpuEllipsoid.h"
#include "gpuLinAlg.h"

using std::vector;
using isce::cuda::core::gpuEllipsoid;
using isce::cuda::core::gpuLinAlg;

CUDA_DEV void gpuEllipsoid::lonLatToXyz(const double *llh, double *xyz) const{
    double re = rEast(llh[1]);
    xyz[0] = (re + llh[2]) * cos(llh[1]) * cos(llh[0]);
    xyz[1] = (re + llh[2]) * cos(llh[1]) * sin(llh[0]);
    xyz[2] = ((re * (1. - e2)) + llh[2]) * sin(llh[1]);
}

CUDA_DEV void gpuEllipsoid::xyzToLonLat(const double *xyz, double *llh) const{
    double p = (pow(xyz[0],2) + pow(xyz[1],2)) / pow(a,2);
    double q = ((1. - e2) * pow(xyz[2],2)) / pow(a,2);
    double r = (p + q - pow(e2,2)) / 6.;
    double s = (pow(e2,2) * p * q) / (4. * pow(r,3));
    double t = cbrt(1. + s + sqrt(s * (2. + s)));
    double u = r * (1. + t + (1. / t));
    double rv = sqrt(pow(u,2) + (pow(e2,2) * q));
    double w = (e2 * (u + rv - q)) / (2. * rv);
    double k = sqrt(u + rv + pow(w,2)) - w;
    double d = (k * sqrt(pow(xyz[0],2) + pow(xyz[1],2))) / (k + e2);
    llh[1] = atan2(xyz[2],d);
    llh[0] = atan2(xyz[1],xyz[0]);
    llh[2] = ((k + e2 - 1.) * sqrt(pow(d,2) + pow(xyz[2],2))) / k;
}

CUDA_DEV void gpuEllipsoid::TCNbasis(double *pos, double *vel, double *t, double *c, double *n) const{
    double temp[3];
    xyzToLonLat(pos,temp);
    n[0] = -cos(temp[0]) * cos(temp[1]);
    n[1] = -cos(temp[0]) * sin(temp[1]);
    n[2] = -sin(temp[0]);
    gpuLinAlg::cross(n,vel,temp);
    gpuLinAlg::unitVec(temp,c);
    gpuLinAlg::cross(c,n,temp);
    gpuLinAlg::unitVec(temp,t);
}

CUDA_GLOBAL void lonLatToXyz_d(gpuEllipsoid elp, const double *llh, double *xyz) {
    /*
     *  GPU-side helper kernel for lonLatToXyz_h to use as a consistency check. Note that elp, llh,
     *  and xyz are GPU-side memory constructs.
     */
    elp.lonLatToXyz(llh, xyz);
}

CUDA_HOST void gpuEllipsoid::lonLatToXyz_h(cartesian_t &llh, cartesian_t &xyz) {
    /*
     *  CPU-side function to call the corresponding GPU function on a single thread. This function
     *  is primarily meant to be used as a consistency check in the test suite, but may be used in
     *  other contexts.
     */
    // Check inputs for valid length
    //checkVecLen(llh,3);
    //checkVecLen(xyz,3);
    // Malloc memory on the GPU and copy the llh inputs over
    double *llh_d, *xyz_d;
    hipMalloc((double**)&llh_d, 3*sizeof(double));
    hipMalloc((double**)&xyz_d, 3*sizeof(double));
    hipMemcpy(llh_d, llh.data(), 3*sizeof(double), hipMemcpyHostToDevice);
    // Run the lonLatToXyz function on the gpuEllipsoid object on the GPU
    dim3 grid(1), block(1);
    lonLatToXyz_d <<<grid,block>>>(*this, llh_d, xyz_d);
    // Copy the resulting xyz back to the CPU-side vector
    hipMemcpy(xyz.data(), xyz_d, 3*sizeof(double), hipMemcpyDeviceToHost);
    hipFree(llh_d);
    hipFree(xyz_d);
}

CUDA_GLOBAL void xyzToLonLat_d(gpuEllipsoid elp, const double *xyz, double *llh) {
    /*
     * GPU-side helper kernel for xyzToLonLat_h to use as a consistency check. Note that elp, xyz,
     * and llh are GPU-side memory constructs.
     */
    elp.xyzToLonLat(xyz, llh);
}


CUDA_HOST void gpuEllipsoid::xyzToLonLat_h(cartesian_t &xyz, cartesian_t &llh) {
    /*
     *  CPU-side function to call the corresponding GPU function on a single thread. This function
     *  is primarily meant to be used as a consistency check in the test suite, but may be used in
     *  other contexts.
     */
     // Check inputs for valid length
     //checkVecLen(xyz,3);
     //checkVecLen(llh,3);
     // Malloc memory on the GPU and copy the xyz inputs over
     double *xyz_d, *llh_d;
     hipMalloc((double**)&xyz_d, 3*sizeof(double));
     hipMalloc((double**)&llh_d, 3*sizeof(double));
     hipMemcpy(xyz_d, xyz.data(), 3*sizeof(double), hipMemcpyHostToDevice);
     // Run the xyzToLonLat function on the gpuEllipsoid object on the GPU
     dim3 grid(1), block(1);
     xyzToLonLat_d <<<grid,block>>>(*this, xyz_d, llh_d);
     // Copy the resulting xyz back to the CPU-side vector
     hipMemcpy(llh.data(), llh_d, 3*sizeof(double), hipMemcpyDeviceToHost);
     hipFree(xyz_d);
     hipFree(llh_d);
}

