#include "hip/hip_runtime.h"
// -*- C++ -*-
// -*- coding: utf-8 -*-
//
// Source Author: Liang Yu
// Copyright 2019

#include "gpuFilter.h"
#include "isce/io/Raster.h"

#include "isce/cuda/hip/hip_runtime_api.h"
#include "isce/cuda/helper_functions.h"

#define THRD_PER_BLOCK 1024 // Number of threads per block (should always %32==0)

using isce::cuda::signal::gpuFilter;

template<class T>
gpuFilter<T>::~gpuFilter()
{
    if (_filter_set) {
        checkCudaErrors(hipFree(_d_filter));
    }
}

// do all calculations in place with data stored on device within signal
template<class T>
void gpuFilter<T>::
filter(gpuSignal<T> &signal)
{
    signal.forward();

    auto n_signal_elements = signal.getNumElements();

    // determine block layout
    dim3 block(THRD_PER_BLOCK);
    dim3 grid((n_signal_elements+(THRD_PER_BLOCK-1))/THRD_PER_BLOCK);

    filter_g<<<grid, block>>>(reinterpret_cast<thrust::complex<T> *>(signal.getDevicePtr()),
            reinterpret_cast<thrust::complex<T> *>(&_d_filter),
            n_signal_elements);

    checkCudaErrors(hipDeviceSynchronize());

    signal.inverse();
}


// pass in device pointer to filter on
template<class T>
void gpuFilter<T>::
filter(thrust::complex<T> *data)
{
    _signal.forwardDevMem(reinterpret_cast<T *>(data));

    auto n_signal_elements = _signal.getNumElements();

    // determine block layout
    dim3 block(THRD_PER_BLOCK);
    dim3 grid((n_signal_elements+(THRD_PER_BLOCK-1))/THRD_PER_BLOCK);

    filter_g<<<grid, block>>>(data,
            reinterpret_cast<thrust::complex<T> *>(_d_filter),
            n_signal_elements);

    checkCudaErrors(hipDeviceSynchronize());

    _signal.inverseDevMem(reinterpret_cast<T *>(data));
}


// pass in host memory to copy to device to be filtered
// interim spectrum is saved as well
template<class T>
void gpuFilter<T>::
filter(std::valarray<std::complex<T>> &signal,
        std::valarray<std::complex<T>> &spectrum)
{
    _signal.dataToDevice(signal);
    _signal.forward();

    // save spectrum
    _signal.dataToHost(spectrum);

    // determine block layout
    dim3 block(THRD_PER_BLOCK);
    dim3 grid((signal.size()+(THRD_PER_BLOCK-1))/THRD_PER_BLOCK);

    filter_g<<<grid, block>>>(reinterpret_cast<thrust::complex<T> *>(_signal.getDevicePtr()),
            reinterpret_cast<thrust::complex<T> *>(&_d_filter),
            signal.size());

    checkCudaErrors(hipDeviceSynchronize());

    _signal.inverse();

    // copy signal to host
    _signal.dataToHost(signal);
}

template<class T>
void gpuFilter<T>::
writeFilter(size_t ncols, size_t nrows)
{
    isce::io::Raster filterRaster("filter.bin", ncols, nrows, 1, GDT_CFloat32, "ENVI");
}

template<class T>
__global__ void phaseShift_g(thrust::complex<T> *slc,
        T *range,
        double pxlSpace,
        T conj,
        double wavelength,
        T wave_div,
        int n_elements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n_elements) {
        T phase = 4.0*M_PI*pxlSpace*range[i]/wavelength;
        thrust::complex<T> complex_phase(cos(phase/wave_div), conj*sin(phase/wave_div));
        slc[i] *= complex_phase;
    }
}

template<>
__global__ void phaseShift_g<float>(thrust::complex<float> *slc,
        float *range,
        double pxlSpace,
        float conj,
        double wavelength,
        float wave_div,
        int n_elements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n_elements) {
        float phase = 4.0*M_PI*pxlSpace*range[i]/wavelength;
        thrust::complex<float> complex_phase(cosf(phase/wave_div), conj*sinf(phase/wave_div));
        slc[i] *= complex_phase;
    }
}

template<class T>
__global__ void filter_g(thrust::complex<T> *signal, thrust::complex<T> *filter, int n_elements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n_elements) {
        signal[i] *= filter[i];
    }
}

template<class T>
__global__ void sumSpectrum_g(thrust::complex<T> *spectrum, T *spectrum_sum, int n_rows, int n_cols)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n_cols) {
        for (int i_row = 0; i_row < n_rows; ++i_row) {
            spectrum_sum[i] += abs(spectrum[i_row*n_cols + i]);
        }
    }
}

// DECLARATIONS
template class gpuFilter<float>;

template __global__ void
sumSpectrum_g<float>(thrust::complex<float> *spectrum, float *spectrum_sum, int n_rows, int n_cols);
