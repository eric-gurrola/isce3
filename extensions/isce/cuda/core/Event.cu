#include <isce/cuda/except/Error.h>

#include "Event.h"

namespace isce { namespace cuda { namespace core {

Event::Event()
{
    // construct shared pointer with custom deleter to clean up the event object
    _event = std::shared_ptr<hipEvent_t> {
        new hipEvent_t,
        [](hipEvent_t * event) noexcept {
            hipEventDestroy(*event);
            delete event;
        }};

    // init event with flags for best runtime performance
    checkCudaErrors( hipEventCreateWithFlags(_event.get(),
            hipEventDisableTiming | hipEventBlockingSync) );
}

bool operator==(Event lhs, Event rhs)
{
    return lhs.get() == rhs.get();
}

bool operator!=(Event lhs, Event rhs)
{
    return !(lhs == rhs);
}

void synchronize(Event event)
{
    checkCudaErrors( hipEventSynchronize(event.get()) );
}

bool query(Event event)
{
    hipError_t status = hipEventQuery(event.get());
    if (status == hipSuccess) {
        return true;
    }
    if (status == hipErrorNotReady) {
        return false;
    }

    // this line should always throw - return statement
    // is just needed to prevent compiler warnings
    checkCudaErrors(status);
    return false;
}

}}}

