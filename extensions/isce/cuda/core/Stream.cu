#include <isce/cuda/except/Error.h>

#include "Stream.h"

namespace isce { namespace cuda { namespace core {

Stream::Stream()
{
    // construct shared pointer with custom deleter to
    // clean up the stream object
    _stream = std::shared_ptr<hipStream_t> {
        new hipStream_t,
        [](hipStream_t * stream) noexcept {
            hipStreamDestroy(*stream);
            delete stream;
        }};

    // init stream
    checkCudaErrors( hipStreamCreate(_stream.get()) );
}

Stream::Stream(std::nullptr_t)
{
    _stream = std::make_shared<hipStream_t>();
}

Stream::operator bool() const
{
    return get();
}

void Stream::record_event(Event event) const
{
    checkCudaErrors( hipEventRecord(event.get(), get()) );
}

Event Stream::record_event() const
{
    Event event;
    record_event(event);
    return event;
}

void Stream::wait_event(Event event) const
{
    checkCudaErrors( hipStreamWaitEvent(get(), event.get(), 0) );
}

bool operator==(Stream lhs, Stream rhs)
{
    return lhs.get() == rhs.get();
}

bool operator!=(Stream lhs, Stream rhs)
{
    return !(lhs == rhs);
}

void synchronize(Stream stream)
{
    checkCudaErrors( hipStreamSynchronize(stream.get()) );
}

bool query(Stream stream)
{
    hipError_t status = hipStreamQuery(stream.get());
    if (status == hipSuccess) {
        return true;
    }
    if (status == hipErrorNotReady) {
        return false;
    }

    // this line should always throw - return statement
    // is just needed to prevent compiler warnings
    checkCudaErrors(status);
    return false;
}

}}}

