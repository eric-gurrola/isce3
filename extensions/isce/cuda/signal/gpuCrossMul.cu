#include "hip/hip_runtime.h"
// -*- C++ -*-
// -*- coding: utf-8 -*-
//
// Author: Liang Yu
// Copyright 2019
//

#include "gpuCrossMul.h"
#include "gpuSignal.h"
#include "gpuLooks.h"
#include "isce/signal/Signal.h"
#include "isce/signal/Filter.h"
#include <isce/cuda/except/Error.h>

// debug includes
#include <fstream>
#include <iostream>
#include <stdio.h>

#define THRD_PER_BLOCK 1024 // Number of threads per block (should always %32==0)

/*
output
    thrust::complex *ifgram (n_cols*n_rows)
input
    thrust::complex *refSlcUp ((oversample*n_ff)t*n_rows)
    thrust::complex *secSlcUp
    int n_rows
    int n_cols
    int n_fft
    int oversample
*/
template <typename T>
__global__ void interferogram_g(thrust::complex<T> *ifgram,
        thrust::complex<T> *refSlcUp,
        thrust::complex<T> *secSlcUp,
        int n_rows,
        int n_cols,
        int n_fft,
        int oversample_i,
        T oversample_f)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // make sure index within ifgram size bounds
    if (i < n_rows * n_cols) {
        auto i_row = i / n_cols;
        auto i_col = i % n_cols;

        ifgram[i] = thrust::complex<T>(0.0, 0.0);
        for (int j = 0; j < oversample_i; ++j) {
            auto ref_val = refSlcUp[i_row*oversample_i*n_fft + i_col];
            auto sec_val_conj = conj(secSlcUp[i_row*oversample_i*n_fft + i_col]);
            ifgram[i] += ref_val * sec_val_conj;
            auto wtf = ref_val * sec_val_conj;
        }
        ifgram[i] /= oversample_f;
    }
}


template <typename T>
__global__ void calculate_coherence_g<T>(T *ref_amp,
        T *sec_amp,
        thrust::complex<T> *coherence,
        int n_elements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // make sure index within ifgram size bounds
    if (i < n_elements) {
        coherence[i] = abs(coherence[i]) / sqrtf(ref_amp[i] * sec_amp[i]);
    }
}


/** Set number of range looks */
void isce::cuda::signal::gpuCrossmul::
rangeLooks(int rngLks) {
    _rangeLooks = rngLks;
    _doMultiLook = true;
}

/** Set number of azimuth looks */
void isce::cuda::signal::gpuCrossmul::
azimuthLooks(int azLks) {
    _azimuthLooks = azLks;
    _doMultiLook = true;
}

void isce::cuda::signal::gpuCrossmul::
doppler(isce::core::LUT1d<double> refDoppler,
        isce::core::LUT1d<double> secDoppler)
{
    _refDoppler = refDoppler;
    _secDoppler = secDoppler;
}


void isce::cuda::signal::gpuCrossmul::
crossmul(isce::io::Raster& referenceSLC,
        isce::io::Raster& secondarySLC,
        isce::io::Raster& interferogram,
        isce::io::Raster& coherence)
{
    _doCommonRangeBandFilter = false;
    isce::io::Raster rngOffsetRaster("/vsimem/dummy", 1, 1, 1, GDT_CFloat32, "ENVI");
    crossmul(referenceSLC,
            secondarySLC,
            rngOffsetRaster,
            interferogram,
            coherence);

}

void isce::cuda::signal::gpuCrossmul::
crossmul(isce::io::Raster& referenceSLC,
        isce::io::Raster& secondarySLC,
        isce::io::Raster& rngOffsetRaster,
        isce::io::Raster& interferogram,
        isce::io::Raster& coherenceRaster)
{
    size_t nrows = referenceSLC.length();
    size_t ncols = referenceSLC.width();

    // setting the parameters of the multi-looking oject
    if (_doMultiLook) {
        // Making sure that the number of rows in each block (blockRows)
        // to be an integer number of azimuth looks.
        blockRows = (blockRows/_azimuthLooks)*_azimuthLooks;
    }

    size_t blockRowsMultiLooked = blockRows/_azimuthLooks;
    size_t ncolsMultiLooked = ncols/_rangeLooks;

    // number of blocks to process
    size_t nblocks = nrows / blockRows;
    if (nblocks == 0) {
        nblocks = 1;
    } else if (nrows % (nblocks * blockRows) != 0) {
        nblocks += 1;
    }

    // signal object for upsampling
    isce::cuda::signal::gpuSignal<float> signalNoUpsample(HIPFFT_C2C);
    isce::cuda::signal::gpuSignal<float> signalUpsample(HIPFFT_C2C);

    // Compute FFT size (power of 2)
    size_t nfft;
    signalNoUpsample.nextPowerOfTwo(ncols, nfft);

    // set upsampling FFT plans
    signalNoUpsample.rangeFFT(nfft, blockRows);
    signalUpsample.rangeFFT(nfft*oversample, blockRows);

    // set not upsampled parameters
    auto n_slc = nfft*blockRows;
    auto slc_size = n_slc * sizeof(thrust::complex<float>);

    // storage for a block of reference SLC data
    std::valarray<std::complex<float>> refSlc(n_slc);
    thrust::complex<float> *d_refSlc;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_refSlc), slc_size));

    // storage for a block of secondary SLC data
    std::valarray<std::complex<float>> secSlc(n_slc);
    thrust::complex<float> *d_secSlc;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_secSlc), slc_size));

    // set upsampled parameters
    auto n_slcUpsampled = oversample * nfft * blockRows;
    auto slcUpsampled_size = n_slcUpsampled * sizeof(thrust::complex<float>);

    // upsampled block of reference SLC
    std::valarray<std::complex<float>> refSlcUpsampled(n_slcUpsampled);
    thrust::complex<float> *d_refSlcUpsampled;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_refSlcUpsampled), slcUpsampled_size));

    // upsampled block of secondary SLC
    std::valarray<std::complex<float>> secSlcUpsampled(n_slcUpsampled);
    thrust::complex<float> *d_secSlcUpsampled;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_secSlcUpsampled), slcUpsampled_size));

    // shift impact
    std::valarray<std::complex<float>> shiftImpact(n_slcUpsampled);
    thrust::complex<float> *d_shiftImpact;
    lookdownShiftImpact(oversample,
            nfft,
            blockRows,
            shiftImpact);
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_shiftImpact), slcUpsampled_size));
    checkCudaErrors(hipMemcpy(d_shiftImpact, &shiftImpact[0], slcUpsampled_size, hipMemcpyHostToDevice));

    // interferogram
    auto n_ifgram = ncols * blockRows;
    auto ifgram_size = n_ifgram * sizeof(thrust::complex<float>);
    std::valarray<std::complex<float>> ifgram(n_ifgram);
    thrust::complex<float> *d_ifgram;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_ifgram), ifgram_size));

    // range offset
    std::valarray<double> rngOffset(ncols*blockRows);
    thrust::complex<double> *d_rngOffset;
    auto rngOffset_size = ncols*nrows*sizeof(double);
    if (_doCommonRangeBandFilter) {
        // only malloc if we're using...
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_rngOffset), rngOffset_size));
    }

    // multilooked products container and parameters
    std::valarray<std::complex<float>> ifgram_mlook(0);
    std::valarray<float> coherence(0);
    int n_mlook = blockRowsMultiLooked*ncolsMultiLooked;
    auto mlook_size = n_mlook*sizeof(float);

    // CUDA device memory allocation
    thrust::complex<float> *d_ifgram_mlook;
    float *d_ref_amp_mlook;
    float *d_sec_amp_mlook;

    if (_doMultiLook) {
        ifgram_mlook.resize(n_mlook);
        coherence.resize(n_mlook);
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_ifgram_mlook), 2*mlook_size)); // 2* because imaginary
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_ref_amp_mlook), mlook_size));
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_sec_amp_mlook), mlook_size));
    }

    // filter objects
    isce::cuda::signal::gpuAzimuthFilter<float> azimuthFilter;
    isce::cuda::signal::gpuRangeFilter<float> rangeFilter;

    // determine block layout
    dim3 block(THRD_PER_BLOCK);
    dim3 grid_hi((refSlc.size()*oversample+(THRD_PER_BLOCK-1))/THRD_PER_BLOCK);
    dim3 grid_reg((refSlc.size()+(THRD_PER_BLOCK-1))/THRD_PER_BLOCK);
    dim3 grid_lo((blockRowsMultiLooked*ncolsMultiLooked+(THRD_PER_BLOCK-1))/THRD_PER_BLOCK);

    // configure azimuth filter
    if (_doCommonAzimuthBandFilter) {
        azimuthFilter.constructAzimuthCommonbandFilter(
                _refDoppler,
                _secDoppler,
                _commonAzimuthBandwidth,
                _prf,
                _beta,
                nfft, 
                blockRows);
    }

    // loop over all blocks
    for (size_t i_block = 0; i_block < nblocks; ++i_block) {
        std::cout << "i_block: " << i_block << std::endl;
        // start row for this block
        size_t rowStart;
        rowStart = i_block * blockRows;

        //number of lines of data in this block. blockRowsData<= blockRows
        //Note that blockRows is fixed number of lines
        //blockRowsData might be less than or equal to blockRows.
        //e.g. if nrows = 512, and blockRows = 100, then
        //blockRowsData for last block will be 12
        size_t blockRowsData;
        if ((rowStart + blockRows) > nrows) {
            blockRowsData = nrows - rowStart;
        } else {
            blockRowsData = blockRows;
        }

        // fill the valarray with zero before getting the block of the data
        refSlc = 0;
        secSlc = 0;
        refSlcUpsampled = 0;
        secSlcUpsampled = 0;
        ifgram = 0;

        // get a block of reference and secondary SLC data
        // and a block of range offsets
        // This will change once we have the functionality to
        // get a block of data directly in to a slice
        std::valarray<std::complex<float>> dataLine(ncols);
        for (size_t line = 0; line < blockRowsData; ++line){
            referenceSLC.getLine(dataLine, rowStart + line);
            refSlc[std::slice(line*nfft, ncols, 1)] = dataLine;
            secondarySLC.getLine(dataLine, rowStart + line);
            secSlc[std::slice(line*nfft, ncols, 1)] = dataLine;
        }
        checkCudaErrors(hipMemcpy(d_refSlc, &refSlc[0], slc_size, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(d_secSlc, &secSlc[0], slc_size, hipMemcpyHostToDevice));

        // apply azimuth filter (do inplace)
        if (_doCommonAzimuthBandFilter) {
            azimuthFilter.filter(d_refSlc);
            azimuthFilter.filter(d_secSlc);
        }

        // apply range filter (do inplace)
        if (_doCommonRangeBandFilter) {
            // Read range offsets
            std::valarray<double> offsetLine(ncols);
            for (size_t line = 0; line < blockRowsData; ++line){
                rngOffsetRaster.getLine(offsetLine, rowStart + line);
                rngOffset[std::slice(line*ncols, ncols, 1)] = offsetLine;
            }
            checkCudaErrors(hipMemcpy(d_rngOffset, &rngOffset[0], rngOffset_size, hipMemcpyHostToDevice));

            rangeFilter.filterCommonRangeBand(
                    reinterpret_cast<float *>(&d_refSlc),
                    reinterpret_cast<float *>(&d_secSlc),
                    reinterpret_cast<float *>(&d_rngOffset));
        }

        // upsample reference and secondary done on device
        upsample(signalNoUpsample,
                signalUpsample,
                d_refSlc,
                d_refSlcUpsampled,
                d_shiftImpact);
        upsample(signalNoUpsample,
                signalUpsample,
                d_secSlc,
                d_secSlcUpsampled,
                d_shiftImpact);

        // run kernels to compute oversampled interforgram
        // refSignal overwritten with upsampled interferogram
        // reduce from nfft*oversample*blockRows to ncols*blockRows
        float oversample_f = float(oversample);
        interferogram_g<<<grid_reg, block>>>(
                d_ifgram,
                d_refSlcUpsampled,
                d_secSlcUpsampled,
                nrows, ncols, nfft, oversample, oversample_f);

        if (_doMultiLook) {

            // reduce ncols*nrow to ncolsMultiLooked*blockRowsMultiLooked
            multilooks_g<<<grid_lo, block>>>(
                    d_ifgram_mlook,
                    d_ifgram,
                    ncols,                          // n columns hi res
                    ncolsMultiLooked,               // n cols lo res
                    _azimuthLooks,                  // col resize factor of hi to lo
                    _rangeLooks,                    // col resize factor of hi to lo
                    n_mlook,                        // number of lo res elements
                    float(_azimuthLooks*_rangeLooks));

            // get data to HOST
            checkCudaErrors(hipMemcpy(&ifgram_mlook[0], d_ifgram_mlook, mlook_size*2, hipMemcpyDeviceToHost));

            interferogram.setBlock(ifgram_mlook, 0, rowStart/_azimuthLooks,
                        ncols/_rangeLooks, blockRowsData/_azimuthLooks);

            // write reduce+abs and set blocks
            multilooks_power_g<<<grid_lo, block>>>(
                    d_ref_amp_mlook,
                    d_refSlc,
                    2,
                    ncols,
                    ncolsMultiLooked,
                    _azimuthLooks,                  // row resize factor of hi to lo
                    _rangeLooks,                    // col resize factor of hi to lo
                    n_mlook,                        // number of lo res elements
                    float(_azimuthLooks*_rangeLooks));

            multilooks_power_g<<<grid_lo, block>>>(
                    d_sec_amp_mlook,
                    d_secSlc,
                    2,
                    ncols,
                    ncolsMultiLooked,
                    _azimuthLooks,                  // row resize factor of hi to lo
                    _rangeLooks,                    // col resize factor of hi to lo
                    n_mlook,                        // number of lo res elements
                    float(_azimuthLooks*_rangeLooks));

            // perform coherence calculation in place overwriting d_ifgram_mlook
            calculate_coherence_g<<<grid_lo, block>>>(d_ref_amp_mlook,
                    d_sec_amp_mlook,
                    d_ifgram_mlook,
                    ifgram_mlook.size());

            // get data to HOST; overwrite multilooked ifgram with multilooked coherence
            checkCudaErrors(hipMemcpy(&ifgram_mlook[0], d_ifgram_mlook, ifgram_mlook.size()*sizeof(float)*2, hipMemcpyDeviceToHost));

            // set blocks accordingly
            coherenceRaster.setBlock(coherence, 0, rowStart/_azimuthLooks,
                        ncols/_rangeLooks, blockRowsData/_azimuthLooks);

        } else {
            // get data to HOST
            checkCudaErrors(hipMemcpy(&ifgram[0], d_ifgram, ifgram_size, hipMemcpyDeviceToHost));

            // set the block of interferogram
            interferogram.setBlock(ifgram, 0, rowStart, ncols, blockRowsData);
        }

    }

    // liberate all device memory
    checkCudaErrors(hipFree(d_refSlc));
    checkCudaErrors(hipFree(d_secSlc));
    checkCudaErrors(hipFree(d_refSlcUpsampled));
    checkCudaErrors(hipFree(d_secSlcUpsampled));
    checkCudaErrors(hipFree(d_shiftImpact));
    checkCudaErrors(hipFree(d_ifgram));
    if (_doCommonRangeBandFilter) {
        checkCudaErrors(hipFree(d_rngOffset));
    }
    if (_doMultiLook) {
        checkCudaErrors(hipFree(d_ifgram_mlook));
        checkCudaErrors(hipFree(d_ref_amp_mlook));
        checkCudaErrors(hipFree(d_sec_amp_mlook));
    }

}


/**
 * @param[in] oversample upsampling factor
 * @param[in] nfft fft length in range direction
 * @param[in] blockRows number of rows of the block of data
 * @param[out] shiftImpact frequency responce (a linear phase) to a sub-pixel shift in time domain introduced by upsampling followed by downsampling
 */
void lookdownShiftImpact(size_t oversample,
        size_t nfft,
        size_t blockRows,
        std::valarray<std::complex<float>> &shiftImpact)
{
    // range frequencies given nfft and oversampling factor
    std::valarray<double> rangeFrequencies(oversample*nfft);

    // sampling interval in range
    double dt = 1.0/oversample;

    // get the vector of range frequencies
    isce::signal::fftfreq(dt, rangeFrequencies);


    // in the process of upsampling the SLCs, creating upsampled interferogram
    // and then looking down the upsampled interferogram to the original size of
    // the SLCs, a shift is introduced in range direction.
    // As an example for a signal with length of 5 and :
    // original sample locations:   0       1       2       3        4
    // upsampled sample locations:  0   0.5 1  1.5  2  2.5  3   3.5  4   4.5
    // Looked dow sample locations:   0.25    1.25    2.25    3.25    4.25
    // Obviously the signal after looking down would be shifted by 0.25 pixel in
    // range comared to the original signal. Since a shift in time domain introduces
    // a liner phase in frequency domain, we compute the impact in frequency domain.

    // the constant shift based on the oversampling factor
    double shift = 0.0;
    shift = (1.0 - 1.0/oversample)/2.0;

    // compute the frequency response of the subpixel shift in range direction
    std::valarray<std::complex<float>> shiftImpactLine(oversample*nfft);
    for (size_t col=0; col<shiftImpactLine.size(); ++col){
        double phase = -1.0*shift*2.0*M_PI*rangeFrequencies[col];
        shiftImpactLine[col] = std::complex<float> (std::cos(phase),
                                                    std::sin(phase));
    }

    // The imapct is the same for each range line. Therefore copying the line for the block
    for (size_t line = 0; line < blockRows; ++line){
            shiftImpact[std::slice(line*nfft*oversample, nfft*oversample, 1)] = shiftImpactLine;
    }
}

/*
forced instantiation
   */

template __global__
void interferogram_g<float>(thrust::complex<float> *ifgram,
        thrust::complex<float> *refSlcUp,
        thrust::complex<float> *secSlcUp,
        int n_rows,
        int n_cols,
        int n_fft,
        int oversample_i,
        float oversample_f);

