#include "hip/hip_runtime.h"
// -*- C++ -*-
// -*- coding: utf-8 -*-
//
// michael a.g. aïvázis <michael.aivazis@para-sim.com>
// parasim
// (c) 1998-2019 all rights reserved
//


// configuration
//#include <portinfo>
// STL
#include <exception>
#include <complex>
#include <string>
// cuda
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
// pyre
#include <pyre/journal.h>
// local declarations
#include "kernels.h"

// helpers
template <std::size_t T, typename value_t = float>
__global__
void
_refStats(value_t * rArena,
          std::size_t refDim, std::size_t cellsPerTilePair,
          value_t * stats);


// compute the amplitude of the signal tiles, assuming pixels are of type std::complex<float>
void
ampcor::cuda::kernels::
refStats(float * rArena,
         std::size_t pairs, std::size_t refDim, std::size_t cellsPerTilePair,
         float * stats)
{
    // make a channel
    pyre::journal::debug_t channel("ampcor.cuda");
    // show me
    channel
        << pyre::journal::at(__HERE__)
        << "arena has " << pairs << " blocks of " << cellsPerTilePair << " cells;"
        << " the reference tiles are " << refDim << "x" << refDim
        << pyre::journal::endl;

    // the number of threads per block
    auto T = refDim;
    // the number of blocks
    auto B = pairs;
    // the amount of shared memory
    auto S = std::max(64ul, T) * sizeof(float);

    // show me
    channel << pyre::journal::at(__HERE__);
    // deploy
    if (refDim <= 32) {
        // show me
        channel << "deploying the 32x32 kernel";
        // with 32x32 tiles
        _refStats<32><<<B, 32, S>>>(rArena, refDim, cellsPerTilePair, stats);
    } else if (refDim <= 64) {
        // show me
        channel << "deploying the 64x64 kernel";
        // with 64x64 tiles
        _refStats<64><<<B, 64, S>>>(rArena, refDim, cellsPerTilePair, stats);
    } else if (refDim <= 128) {
        // show me
        channel << "deploying the 128x128 kernel";
        // with 128x128 tiles
        _refStats<128><<<B, 128, S>>>(rArena, refDim, cellsPerTilePair, stats);
    } else if (refDim <= 256) {
        // show me
        channel << "deploying the 256x256 kernel";
        // with 256x256 tiles
        _refStats<256><<<B, 256, S>>>(rArena, refDim, cellsPerTilePair, stats);
    } else if (refDim <= 512) {
        // show me
        channel << "deploying the 512x512 kernel";
        // with 512x512 tiles
        _refStats<512><<<B, 512, S>>>(rArena, refDim, cellsPerTilePair, stats);
    } else {
        // complain
        throw std::runtime_error("cannot handle reference tiles of this shape");
    }
    // flush
    channel << pyre::journal::endl;

    // wait for the device to finish
    hipError_t status = hipDeviceSynchronize();
    // if something went wrong
    if (status != hipSuccess) {
        // form the error description
        std::string description = hipGetErrorName(status);
        // make a channel
        pyre::journal::error_t channel("ampcor.cuda");
        // complain
        channel
            << pyre::journal::at(__HERE__)
            << "while ensuring the detected reference tiles have zero mean: "
            << description << " (" << status << ")"
            << pyre::journal::endl;
        // bail
        throw std::runtime_error(description);
    }

    // all done
    return;
}


// implementations
template <std::size_t T, typename value_t>
__global__
void
_refStats(value_t * rArena,
          std::size_t refDim, std::size_t cellsPerTilePair,
          value_t * stats)
{
    // build the workload descriptors
    // global
    // std::size_t B = gridDim.x;      // number of blocks
    // std::size_t T = blockDim.x;     // number of threads per block
    // auto W = B*T;                   // total number of workers
    // local
    std::size_t b = blockIdx.x;        // my block id
    std::size_t t = threadIdx.x;       // my thread id
    // auto w = b*T + t;               // my worker id

    // N.B.: do not be tempted to terminate early threads that have no assigned workload; their
    // participation is required to make sure that shared memory is properly zeored out for the
    // nominally out of bounds accesses

    // access to my shared memory
    extern __shared__ value_t scratch[];
    // handle to my thread block group
    cooperative_groups::thread_block cta = cooperative_groups::this_thread_block();

    // step one: every thread sums a column of its tile
    // find the start of my tile by skipping the tile pairs handled by the lesser blocks
    auto tile = rArena + b*cellsPerTilePair;
    // compute the location of the cell past the end of my tile
    auto eot = tile + refDim*refDim;
    // initialize the accumulator
    value_t sum = 0;
    // if my thread id is less than the number of columns, i need to sum up the values;
    // otherwise, my contribution is to zero out my slot in shared memory
    if (t < refDim) {
        // run down my column
        for (auto cell = tile + t; cell < eot; cell += refDim) {
            // picking up contributions
            sum += *cell;
        }
    }
    // store the partial sum in my slot in shared memory
    scratch[t] = sum;
    // make sure everybody is done
    cta.sync();

    // step two: reduction in shared memory
    // for progressively smaller block sizes, the bottom half of the threads collect partial sums
    // N.B.: T is a template parameter, known at compile time, so it's easy for the optimizer to
    // eliminate the impossible clauses
    // for 512 threads per block
    if (T >= 512 && t < 256) {
        // update my partial sum by reading my sibling's value
        sum += scratch[t + 256];
        // and make it available in my shared memory slot
        scratch[t] = sum;
    }
    // make sure everybody is done
    cta.sync();
    // for 256 threads per block
    if (T >= 256 && t < 128) {
        // update my partial sum by reading my sibling's value
        sum += scratch[t + 128];
        // and make it available in my shared memory slot
        scratch[t] = sum;
    }
    // make sure everybody is done
    cta.sync();
    // for 128 threads per block
    if (T >= 128 && t < 64) {
        // update my partial sum by reading my sibling's value
        sum += scratch[t + 64];
        // and make it available in my shared memory slot
        scratch[t] = sum;
    }
    // make sure everybody is done
    cta.sync();
    // on recent architectures, there is a faster way to do the reduction once we reach the
    // warp level; the only cost is that we have to make sure there is enough memory for 64
    // threads, i.e. the shared memory size is bound from below by 64*sizeof(value_t)
    if (t < 32) {
        // if we need to
        if (T >= 64) {
            // update the partial sum from the second warp
            sum += scratch[t + 32];
        }

        // grab the block of active threads
        cooperative_groups::coalesced_group active = cooperative_groups::coalesced_threads();

        // the threads with power of 2 ids
        for (auto offset = 16; offset > 0; offset >>= 1) {
            // reduce using a warp shuffle
            sum += active.shfl_down(sum, offset);
        }
    }
    // finally, thread 0
    if (t == 0) {
        // saves the final value
        scratch[0] = sum / (refDim*refDim);
    }
    // make sure everybody is done
    cta.sync();

    // step three: revisit the tile and subtract this value from all cells, and accumulate the
    // sum of the squares of the resulting cells so we can compute the variance; again, do not
    // be tempted to send idle threads home
    // initialize the variance
    value_t sumsq = 0;
    // only threads assigned to columns do any work
    if (t < refDim) {
        // read the mean value from shared memory
        auto mean = scratch[0];
        // run down my column
        for (auto cell = tile + t; cell < eot; cell += refDim) {
            // get the cell value and subtract the mean
            auto value = *cell - mean;
            // store it
            *cell = value;
            // update the sum of the squares
            sumsq += value*value;
        }
    }
    // store the partial sum in my slot in shared memory
    scratch[t] = sumsq;
    // make sure everybody is done
    cta.sync();

    // step four: reduce the sum of the squares to compute the variance
    // for 512 threads per block
    if (T >= 512 && t < 256) {
        // update my partial sum by reading my sibling's value
        sumsq += scratch[t + 256];
        // and make it available in my shared memory slot
        scratch[t] = sumsq;
    }
    // make sure everybody is done
    cta.sync();
    // for 256 threads per block
    if (T >= 256 && t < 128) {
        // update my partial sum by reading my sibling's value
        sumsq += scratch[t + 128];
        // and make it available in my shared memory slot
        scratch[t] = sumsq;
    }
    // make sure everybody is done
    cta.sync();
    // for 128 threads per block
    if (T >= 128 && t < 64) {
        // update my partial sum by reading my sibling's value
        sumsq += scratch[t + 64];
        // and make it available in my shared memory slot
        scratch[t] = sumsq;
    }
    // make sure everybody is done
    cta.sync();
    // on recent architectures, there is a faster way to do the reduction once we reach the
    // warp level; the only cost is that we have to make sure there is enough memory for 64
    // threads, i.e. the shared memory size is bound from below by 64*sizeof(value_t)
    if (t < 32) {
        // if we need to
        if (T >= 64) {
            // update the partial sum from the second warp
            sumsq += scratch[t + 32];
        }

        // grab the block of active threads
        cooperative_groups::coalesced_group active = cooperative_groups::coalesced_threads();

        // the threads with power of 2 ids
        for (auto offset = 16; offset > 0; offset >>= 1) {
            // reduce using a warp shuffle
            sumsq += active.shfl_down(sumsq, offset);
        }
    }
    // finally, thread 0
    if (t == 0) {
        // computes the variance
        auto var = std::sqrt(sumsq);
        // and saves it in the output array; recall: one block per tile, so the correct memory
        // location to save the answer is given by my block id
        stats[b] = var;
    }

    // all done
    return;
}


// end of file
