#include "hip/hip_runtime.h"
//
// Author: Bryan Riel
// Copyright 2018
//

#include "gpuGeometry.h"
#include <isce/cuda/except/Error.h>
#include <isce/core/Basis.h>

using isce::core::Basis;
using isce::core::Vec3;

/** @param[in] pixel Pixel object
 * @param[in] TCNbasis Geocentric TCN basis corresponding to pixel
 * @param[in] pos/vel state vector position/velocity vectors
 * @param[in] ellipsoid Ellipsoid object
 * @param[in] demInterp gpuDEMInterpolator object
 * @param[out] targetLLH output Lon/Lat/Hae corresponding to pixel
 * @param[in] side +1 for left and -1 for right
 * @param[in] threshold Distance threshold for convergence
 * @param[in] maxIter Number of primary iterations
 * @param[in] extraIter Number of secondary iterations
 *
 * This is the elementary device-side transformation from radar geometry to map geometry. The transformation is applicable for a single slant range and azimuth time. The slant range and Doppler information are encapsulated in the Pixel object, so this function can work for both zero and native Doppler geometries. The azimuth time information is encapsulated in the TCNbasis of the platform. For algorithmic details, see \ref overview_geometry "geometry overview".*/
CUDA_DEV
int isce::cuda::geometry::
rdr2geo(const isce::core::Pixel & pixel,
        const Basis& TCNbasis,
        const Vec3& pos, const Vec3& vel,
        const isce::core::Ellipsoid& ellipsoid,
        const gpuDEMInterpolator & demInterp,
        Vec3& targetLLH,
        int side, double threshold, int maxIter, int extraIter) {

    // Initialization
    Vec3 targetLLH_old, targetVec_old, lookVec;

    // Compute normalized velocity
    const Vec3 vhat = vel.unitVec();

    // Unpack TCN basis vectors to pointers
    const auto& that = TCNbasis.x0();
    const auto& chat = TCNbasis.x1();
    const auto& nhat = TCNbasis.x2();

    // Pre-compute TCN vector products
    const double ndotv = nhat.dot(vhat);
    const double vdott = vhat.dot(that);

    // Compute major and minor axes of ellipsoid
    const double major = ellipsoid.a();
    const double minor = major * std::sqrt(1.0 - ellipsoid.e2());

    // Set up orthonormal system right below satellite
    const double satDist = pos.norm();
    const double eta = 1.0 / std::sqrt(
        std::pow(pos[0] / major, 2) +
        std::pow(pos[1] / major, 2) +
        std::pow(pos[2] / minor, 2)
    );
    const double radius = eta * satDist;
    const double hgt = (1.0 - eta) * satDist;

    // Iterate
    int converged = 0;
    double zrdr = targetLLH[2];
    for (int i = 0; i < (maxIter + extraIter); ++i) {

        // Near nadir test
        if ((hgt - zrdr) >= pixel.range())
            break;

        // Cache the previous solution
        for (int k = 0; k < 3; ++k) {
            targetLLH_old[k] = targetLLH[k];
        }

        // Compute angles
        const double a = satDist;
        const double b = radius + zrdr;
        const double costheta = 0.5 * (a / pixel.range() + pixel.range() / a
                              - (b/a) * (b/pixel.range()));
        const double sintheta = std::sqrt(1.0 - costheta*costheta);

        // Compute TCN scale factors
        const double gamma = pixel.range() * costheta;
        const double alpha = (pixel.dopfact() - gamma * ndotv) / vdott;
        const double beta = -side * std::sqrt(std::pow(pixel.range(), 2)
                                            * std::pow(sintheta, 2)
                                            - std::pow(alpha, 2));

        // Compute vector from satellite to ground
        const Vec3 delta = alpha * that + beta * chat + gamma * nhat;
        Vec3 targetVec = pos + delta;

        // Compute LLH of ground point
        ellipsoid.xyzToLonLat(targetVec, targetLLH);

        // Interpolate DEM at current lat/lon point
        targetLLH[2] = demInterp.interpolateLonLat(targetLLH[0], targetLLH[1]);

        // Convert back to XYZ with interpolated height
        ellipsoid.lonLatToXyz(targetLLH, targetVec);
        // Compute updated target height
        zrdr = targetVec.norm() - radius;

        // Check convergence
        lookVec = pos - targetVec;
        const double rdiff = pixel.range() - lookVec.norm();
        if (std::abs(rdiff) < threshold) {
            converged = 1;
            break;
        // May need to perform extra iterations
        } else if (i > maxIter) {
            // XYZ position of old solution
            ellipsoid.lonLatToXyz(targetLLH_old, targetVec_old);
            // XYZ position of updated solution
            for (int idx = 0; idx < 3; ++idx)
                targetVec[idx] = 0.5 * (targetVec_old[idx] + targetVec[idx]);
            // Repopulate lat, lon, z
            ellipsoid.xyzToLonLat(targetVec, targetLLH);
            // Compute updated target height
            zrdr = targetVec.norm() - radius;
        }
    }

    // ----- Final computation: output points exactly at range pixel if converged

    // Compute angles
    const double a = satDist;
    const double b = radius + zrdr;
    const double costheta = 0.5 * (a / pixel.range() + pixel.range() / a
                          - (b/a) * (b/pixel.range()));
    const double sintheta = std::sqrt(1.0 - costheta*costheta);

    // Compute TCN scale factors
    const double gamma = pixel.range() * costheta;
    const double alpha = (pixel.dopfact() - gamma * ndotv) / vdott;
    const double beta = -side * std::sqrt(std::pow(pixel.range(), 2)
                                        * std::pow(sintheta, 2)
                                        - std::pow(alpha, 2));

    // Compute vector from satellite to ground
    const Vec3 delta = alpha * that + beta * chat + gamma * nhat;
    const Vec3 targetVec = pos + delta;

    // Compute LLH of ground point
    targetLLH = ellipsoid.xyzToLonLat(targetVec);

    // Return convergence flag
    return converged;
}

/*
 * This is a helper function for calling the more primitive version of rdr2geo.
 * For more information, see its CPU equivalent in isce/geometry/geometry.h.
 * This GPU version is simplified since it cannot perform error checking.
 */
__device__ int isce::cuda::geometry::rdr2geo(
        double aztime, double slant_range, double doppler,
        const isce::cuda::core::gpuOrbit& orbit,
        const isce::core::Ellipsoid& ellipsoid,
        const isce::cuda::geometry::gpuDEMInterpolator& dem_interp,
        Vec3& target_llh, double wvl, int side, double threshold,
        int max_iter, int extra_iter) {

    /*
     * Interpolate Orbit to azimuth time, compute TCN basis,
     * and estimate geographic coordinates.
     */

    // Interpolate orbit to get state vector
    Vec3 pos, vel;
    orbit.interpolateWGS84Orbit(aztime, pos.data(), vel.data());

    // Set up geocentric TCN basis
    const Basis tcn_basis(pos, vel);

    // Compute satellite velocity magnitude
    const double vmag = vel.norm();

    // Compute Doppler factor
    const double dopfact = 0.5 * wvl * doppler * slant_range / vmag;

    // Wrap range and Doppler factor in a Pixel object
    isce::core::Pixel pixel(slant_range, dopfact, 0);

    // Finally, call rdr2geo
    return rdr2geo(pixel, tcn_basis, pos, vel, ellipsoid, dem_interp,
                   target_llh, side, threshold, max_iter, extra_iter);
}

/** @param[in] inputLLH Lon/Lat/Hae of target of interest
 * @param[in] ellipsoid Ellipsoid object
 * @param[in] orbit gpuOrbit object
 * @param[in] doppler gpuLUT1d Doppler model
 * @param[out] aztime azimuth time of inputLLH w.r.t reference epoch of the orbit
 * @param[out] slantRange slant range to inputLLH
 * @param[in] wavelength Radar wavelength
 * @param[in] threshold azimuth time convergence threshold in seconds
 * @param[in] maxIter Maximum number of Newton-Raphson iterations
 * @param[in] deltaRange step size used for computing derivative of doppler
 *
 * This is the elementary device-side transformation from map geometry to radar geometry. The transformation is applicable for a single lon/lat/h coordinate (i.e., a single point target). For algorithmic details, see \ref overview_geometry "geometry overview".*/
CUDA_DEV
int isce::cuda::geometry::
geo2rdr(const Vec3& inputLLH,
        const isce::core::Ellipsoid& ellipsoid,
        const isce::cuda::core::gpuOrbit& orbit,
        const isce::cuda::core::gpuLUT1d<double> & doppler,
        double * aztime_result, double * slantRange_result,
        double wavelength, double threshold, int maxIter, double deltaRange) {

    // Cartesian type local variables
    // Temp local variables for results
    double aztime, slantRange;

    // Convert LLH to XYZ
    const Vec3 inputXYZ = ellipsoid.lonLatToXyz(inputLLH);

    // Pre-compute scale factor for doppler
    const double dopscale = 0.5 * wavelength;

    // Use mid-orbit epoch as initial guess
    aztime = orbit.UTCtime[orbit.nVectors / 2];

    // Begin iterations
    int converged = 0;
    double slantRange_old = 0.0;
    for (int i = 0; i < maxIter; ++i) {

        // Interpolate the orbit to current estimate of azimuth time
        Vec3 pos, vel;
        orbit.interpolateWGS84Orbit(aztime, &pos[0], &vel[0]);

        // Compute slant range from satellite to ground point
        const Vec3 dr = inputXYZ - pos;
        slantRange = dr.norm();
        // Check convergence
        if (std::abs(slantRange - slantRange_old) < threshold) {
            converged = 1;
            *slantRange_result = slantRange;
            *aztime_result = aztime;
            return converged;
        } else {
            slantRange_old = slantRange;
        }

        // Compute doppler
        const double dopfact = dr.dot(vel);
        const double fdop = doppler.eval(slantRange) * dopscale;
        // Use forward difference to compute doppler derivative
        const double fdopder = (doppler.eval(slantRange + deltaRange) * dopscale - fdop)
                             / deltaRange;

        // Evaluate cost function and its derivative
        const double fn = dopfact - fdop * slantRange;
        const double c1 = -vel.dot(vel);
        const double c2 = (fdop / slantRange) + fdopder;
        const double fnprime = c1 + c2 * dopfact;

        // Update guess for azimuth time
        aztime -= fn / fnprime;
    }

    // If we reach this point, no convergence for specified threshold
    *slantRange_result = slantRange;
    *aztime_result = aztime;
    return converged;

}

// Create ProjectionBase pointer on the device (meant to be run by a single thread)
__global__
void
createProjection(isce::cuda::core::ProjectionBase ** proj, int epsgCode) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        (*proj) = isce::cuda::core::createProj(epsgCode);
    }
}

// Delete ProjectionBase pointer on the device (meant to be run by a single thread)
__global__
void
deleteProjection(isce::cuda::core::ProjectionBase ** proj) {
    delete *proj;
}

// Helper kernel to call device-side rdr2geo
__global__
void rdr2geo_d(const isce::core::Pixel pixel,
               const Basis TCNbasis,
               const Vec3 pos, const Vec3 vel,
               const isce::core::Ellipsoid ellipsoid,
               isce::cuda::geometry::gpuDEMInterpolator demInterp,
               Vec3* targetLLH,
               int side, double threshold, int maxIter, int extraIter,
               int *resultcode) {

    // Call device function
    *resultcode = isce::cuda::geometry::rdr2geo(
        pixel, TCNbasis, pos, vel, ellipsoid, demInterp, *targetLLH, side,
        threshold, maxIter, extraIter);

}

// Host radar->geo to test underlying functions in a single-threaded context
CUDA_HOST
int isce::cuda::geometry::
rdr2geo_h(const isce::core::Pixel & pixel,
          const Basis& basis,
          const Vec3& pos, const Vec3& vel,
          const isce::core::Ellipsoid & ellipsoid,
          isce::geometry::DEMInterpolator & demInterp,
          Vec3& llh,
          int side, double threshold, int maxIter, int extraIter) {

    // Make GPU objects
    isce::cuda::geometry::gpuDEMInterpolator gpu_demInterp(demInterp);

    // Allocate device memory
    Vec3* llh_d;
    int * resultcode_d;
    hipMalloc((double **) &llh_d, 3*sizeof(double));
    hipMalloc((int **) &resultcode_d, sizeof(int));

    // Copy initial values
    hipMemcpy(llh_d, llh.data(), 3*sizeof(double), hipMemcpyHostToDevice);

    // DEM interpolator initializes its projection and interpolator
    gpu_demInterp.initProjInterp();

    // Run the rdr2geo on the GPU
    dim3 grid(1), block(1);
    rdr2geo_d<<<grid, block>>>(pixel, basis, pos, vel, ellipsoid,
                               gpu_demInterp, llh_d, side, threshold, maxIter,
                               extraIter, resultcode_d);

    // Check for any kernel errors
    checkCudaErrors(hipPeekAtLastError());

    // Delete projection pointer on device
    gpu_demInterp.finalizeProjInterp();

    // Copy the resulting llh back to the CPU
    int resultcode;
    checkCudaErrors(hipMemcpy(llh.data(), llh_d, 3*sizeof(double), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(&resultcode, resultcode_d, sizeof(int), hipMemcpyDeviceToHost));

    // Free memory
    checkCudaErrors(hipFree(llh_d));
    checkCudaErrors(hipFree(resultcode_d));

    // Return result code
    return resultcode;
}

// Helper kernel to call device-side geo2rdr
__global__
void geo2rdr_d(const Vec3 llh,
               isce::core::Ellipsoid ellps,
               isce::cuda::core::gpuOrbit orbit,
               isce::cuda::core::gpuLUT1d<double> doppler,
               double * aztime, double * slantRange,
               double wavelength, double threshold, int maxIter, double deltaRange,
               int *resultcode) {

    // Call device function
    *resultcode = isce::cuda::geometry::geo2rdr(
        llh, ellps, orbit, doppler, aztime, slantRange, wavelength, threshold,
        maxIter, deltaRange);
}

// Host geo->radar to test underlying functions in a single-threaded context
CUDA_HOST
int isce::cuda::geometry::
geo2rdr_h(const cartesian_t& llh,
          const isce::core::Ellipsoid & ellps,
          const isce::core::Orbit & orbit,
          const isce::core::LUT1d<double> & doppler,
          double & aztime, double & slantRange,
          double wavelength, double threshold, int maxIter, double deltaRange) {

    // Make GPU objects
    isce::core::Ellipsoid gpu_ellps(ellps);
    isce::cuda::core::gpuOrbit gpu_orbit(orbit);
    isce::cuda::core::gpuLUT1d<double> gpu_doppler(doppler);

    // Allocate necessary device memory
    double *llh_d, *aztime_d, *slantRange_d;
    int *resultcode_d;
    hipMalloc((double **) &llh_d, 3*sizeof(double));
    hipMalloc((double **) &aztime_d, sizeof(double));
    hipMalloc((double **) &slantRange_d, sizeof(double));
    hipMalloc((int **) &resultcode_d, sizeof(int));

    // Copy input values
    hipMemcpy(llh_d, llh.data(), 3*sizeof(double), hipMemcpyHostToDevice);

    // Run geo2rdr on the GPU
    dim3 grid(1), block(1);
    geo2rdr_d<<<grid, block>>>(llh, gpu_ellps, gpu_orbit, gpu_doppler, aztime_d, slantRange_d,
                               wavelength, threshold, maxIter, deltaRange, resultcode_d);

    // Copy results to CPU and return any error code
    int resultcode;
    hipMemcpy(&aztime, aztime_d, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&slantRange, slantRange_d, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&resultcode, resultcode_d, sizeof(int), hipMemcpyDeviceToHost);

    // Free memory
    hipFree(llh_d);
    hipFree(aztime_d);
    hipFree(slantRange_d);
    hipFree(resultcode_d);

    // Return error code
    return resultcode;
}
