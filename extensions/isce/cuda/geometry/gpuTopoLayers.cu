#include "gpuTopoLayers.h"
#include <isce/cuda/except/Error.h>

namespace isce { namespace cuda { namespace geometry {
    CUDA_HOST gpuTopoLayers::gpuTopoLayers(const isce::geometry::TopoLayers & layers) :
        _length(layers.length()), _width(layers.width()), _owner(true) {

        // Allocate memory
        _nbytes_double = _length * _width * sizeof(double);
        _nbytes_float = _length * _width * sizeof(float);
        checkCudaErrors(hipMalloc((double **) &_x, _nbytes_double));
        checkCudaErrors(hipMalloc((double **) &_y, _nbytes_double));
        checkCudaErrors(hipMalloc((double **) &_z, _nbytes_double));
        checkCudaErrors(hipMalloc((float **) &_inc, _nbytes_float));
        checkCudaErrors(hipMalloc((float **) &_hdg, _nbytes_float));
        checkCudaErrors(hipMalloc((float **) &_localInc, _nbytes_float));
        checkCudaErrors(hipMalloc((float **) &_localPsi, _nbytes_float));
        checkCudaErrors(hipMalloc((float **) &_sim, _nbytes_float));
        checkCudaErrors(hipMalloc((double **) &_crossTrack, _nbytes_double));
    }

    // Destructor
    gpuTopoLayers::~gpuTopoLayers() {
        if (_owner) {
            checkCudaErrors(hipFree(_x));
            checkCudaErrors(hipFree(_y));
            checkCudaErrors(hipFree(_z));
            checkCudaErrors(hipFree(_inc));
            checkCudaErrors(hipFree(_hdg));
            checkCudaErrors(hipFree(_localInc));
            checkCudaErrors(hipFree(_localPsi));
            checkCudaErrors(hipFree(_sim));
            checkCudaErrors(hipFree(_crossTrack));
        }
    }

    // Copy results to host TopoLayers
    CUDA_HOST void gpuTopoLayers::copyToHost(isce::geometry::TopoLayers & layers) {
        checkCudaErrors(hipMemcpy(&layers.x()[0], _x, _nbytes_double,
                        hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(&layers.y()[0], _y, _nbytes_double,
                        hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(&layers.z()[0], _z, _nbytes_double,
                        hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(&layers.inc()[0], _inc, _nbytes_float,
                        hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(&layers.hdg()[0], _hdg, _nbytes_float,
                        hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(&layers.localInc()[0], _localInc, _nbytes_float,
                        hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(&layers.localPsi()[0], _localPsi, _nbytes_float,
                        hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(&layers.sim()[0], _sim, _nbytes_float,
                        hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(&layers.crossTrack()[0], _crossTrack, _nbytes_double,
                        hipMemcpyDeviceToHost));
    }
} } }
